#include "hip/hip_runtime.h"
/*
!
! SEISMIC_CPML Version 1.1.1, November 2009.
!
! Copyright Universite de Pau et des Pays de l'Adour, CNRS and INRIA, France.
! Contributor: Dimitri Komatitsch, komatitsch aT lma DOT cnrs-mrs DOT fr
!
! This software is a computer program whose purpose is to solve
! the three-dimensional isotropic elastic wave equation
! using a finite-difference method with Convolutional Perfectly Matched
! Layer (C-PML) conditions.
!
! This software is a computer program whose purpose is to solve
! the two-dimensional viscoelastic anisotropic or poroelastic wave equation
! using a spectral-element method (SEM).
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 2 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
! The full text of the license is available in file "LICENSE".

  program seismic_CPML_3D_iso_MPI_OpenMP

! 3D elastic finite-difference code in velocity and stress formulation
! with Convolutional-PML (C-PML) absorbing conditions.

! Dimitri Komatitsch, University of Pau, France, April 2007.

! The second-order staggered-grid formulation of Madariaga (1976) and Virieux (1986) is used.

! The C-PML implementation is based in part on formulas given in Roden and Gedney (2000).
!
! Parallel implementation based on both MPI and OpenMP.
! Type for instance "setenv OMP_NUM_THREADS 4" before running in OpenMP if you want 4 tasks.

! The C-PML implementation is based in part on formulas given in Roden and Gedney (2000).
! If you use this code for your own research, please cite some (or all) of these
! articles:
!
! @ARTICLE{MaKoEz08,
! author = {Roland Martin and Dimitri Komatitsch and Abdela\^aziz Ezziani},
! title = {An unsplit convolutional perfectly matched layer improved at grazing
! incidence for seismic wave equation in poroelastic media},
! journal = {Geophysics},
! year = {2008},
! volume = {73},
! pages = {T51-T61},
! number = {4},
! doi = {10.1190/1.2939484}}
!
! @ARTICLE{MaKo09,
! author = {Roland Martin and Dimitri Komatitsch},
! title = {An unsplit convolutional perfectly matched layer technique improved
! at grazing incidence for the viscoelastic wave equation},
! journal = {Geophysical Journal International},
! year = {2009},
! volume = {179},
! pages = {333-344},
! number = {1},
! doi = {10.1111/j.1365-246X.2009.04278.x}}
!
! @ARTICLE{MaKoGe08,
! author = {Roland Martin and Dimitri Komatitsch and Stephen D. Gedney},
! title = {A variational formulation of a stabilized unsplit convolutional perfectly
! matched layer for the isotropic or anisotropic seismic wave equation},
! journal = {Computer Modeling in Engineering and Sciences},
! year = {2008},
! volume = {37},
! pages = {274-304},
! number = {3}}
!
! @ARTICLE{KoMa07,
! author = {Dimitri Komatitsch and Roland Martin},
! title = {An unsplit convolutional {P}erfectly {M}atched {L}ayer improved
!          at grazing incidence for the seismic wave equation},
! journal = {Geophysics},
! year = {2007},
! volume = {72},
! number = {5},
! pages = {SM155-SM167},
! doi = {10.1190/1.2757586}}
!
! The original CPML technique for Maxwell's equations is described in:
!
! @ARTICLE{RoGe00,
! author = {J. A. Roden and S. D. Gedney},
! title = {Convolution {PML} ({CPML}): {A}n Efficient {FDTD} Implementation
!          of the {CFS}-{PML} for Arbitrary Media},
! journal = {Microwave and Optical Technology Letters},
! year = {2000},
! volume = {27},
! number = {5},
! pages = {334-339},
! doi = {10.1002/1098-2760(20001205)27:5<334::AID-MOP14>3.0.CO;2-A}}

! To display the results as color images in the selected 2D cut plane, use:
!
!   " display image*.gif " or " gimp image*.gif "
!
! or
!
!   " montage -geometry +0+3 -rotate 90 -tile 1x21 image*Vx*.gif allfiles_Vx.gif "
!   " montage -geometry +0+3 -rotate 90 -tile 1x21 image*Vy*.gif allfiles_Vy.gif "
!   then " display allfiles_Vx.gif " or " gimp allfiles_Vx.gif "
!   then " display allfiles_Vy.gif " or " gimp allfiles_Vy.gif "
!

! IMPORTANT : all our CPML codes work fine in single precision as well (which is significantly faster).
!             If you want you can thus force automatic conversion to single precision at compile time
!             or change all the declarations and constants in the code from double precision to single.
*/

/*
24-Oct-2015 
conversion from Fortran90 to C++.
replacing openMP and MPI parallelization into CUDA C++ in a single CPU unit.
display results are not done yet
jatmikatejasukmana@gmail.com
*/
/*
3d elastic wave propagation in isotropic medium
Coalesced array computation
*/

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <assert.h>
#include "conio.h"
#include <iomanip>

using namespace std;

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void kersigmaxyz(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *cp, float *cs, float *rho, float *DELTAT, int *DDIMX, int *DDIMY, int *DDIMZ, float *memory_dvx_dx, float *memory_dvy_dy, float *memory_dvz_dz, float *a_x_half, float *a_y, float *a_z, float *b_x_half, float *b_y, float *b_z, float *K_x_half, float *K_y, float *K_z, float *sigmaxx, float *sigmayy, float *sigmazz, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vx, float *vy, float *vz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMZ[0];
	int right = offset + 1;
	int ybottom = offset - DDIMX[0];
	int zbottom = offset - DDIMX[0] * DDIMY[0];
	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;


	if ((index_z >= 2) && (index_z <= DDIMZ[0])) {
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float vp = cp[offset];
				float vs = cs[offset];
				float rhos = rho[offset];

				float lambda = rhos*(vp*vp - 2 * vs*vs);
				float lambdaplus2mu = rhos*vp*vp;

				float DELTAT_lambdaplus2mu = DELTAT[0] * lambdaplus2mu;
				float DELTAT_lambda = DELTAT[0] * lambda;

				float value_dvx_dx = (vx[right] - vx[offset])*ONE_OVER_DELTAX[0];
				float value_dvy_dy = (vy[offset] - vy[ybottom])*ONE_OVER_DELTAY[0];
				float value_dvz_dz = (vz[offset] - vz[zbottom])*ONE_OVER_DELTAZ[0];

				memory_dvx_dx[offset] = b_x_half[iglobal] * memory_dvx_dx[offset] + a_x_half[iglobal] * value_dvx_dx;
				memory_dvy_dy[offset] = b_y[jglobal] * memory_dvy_dy[offset] + a_y[jglobal] * value_dvy_dy;
				memory_dvz_dz[offset] = b_z[kglobal] * memory_dvz_dz[offset] + a_z[kglobal] * value_dvz_dz;

				value_dvx_dx = value_dvx_dx / K_x_half[iglobal] + memory_dvx_dx[offset];
				value_dvy_dy = value_dvy_dy / K_y[jglobal] + memory_dvy_dy[offset];
				value_dvz_dz = value_dvz_dz / K_z[kglobal] + memory_dvz_dz[offset];

				sigmaxx[offset] = DELTAT_lambdaplus2mu * value_dvx_dx + DELTAT_lambda * (value_dvy_dy + value_dvz_dz) + sigmaxx[offset];
				sigmayy[offset] = DELTAT_lambda * (value_dvx_dx + value_dvz_dz) + DELTAT_lambdaplus2mu * value_dvy_dy + sigmayy[offset];
				sigmazz[offset] = DELTAT_lambda * (value_dvx_dx + value_dvy_dy) + DELTAT_lambdaplus2mu * value_dvz_dz + sigmazz[offset];
			}
		}
	}

}

__global__ void kersigmaxy(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *cp, float *cs, float *rho, int *DDIMX, int *DDIMY, int *DDIMZ, float *DELTAT, float *memory_dvy_dx, float *memory_dvx_dy, float *a_x, float *a_y_half, float *b_x, float *b_y_half, float *K_x, float *K_y_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *vx, float *vy, float *sigmaxy) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMZ[0];
	int left = offset - 1;
	int ytop = offset + DDIMX[0];
	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;


	if ((index_z >= 1) && (index_z <= DDIMZ[0])) {
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float vs = (cs[left] + cs[ytop]) / 2;
				float rhos = (rho[left] + rho[ytop]) / 2;

				float mu = rhos*vs*vs;

				float DELTAT_mu = DELTAT[0] * mu;

				float value_dvy_dx = (vy[offset] - vy[left])*ONE_OVER_DELTAX[0];
				float value_dvx_dy = (vx[ytop] - vx[offset])*ONE_OVER_DELTAY[0];

				memory_dvy_dx[offset] = b_x[iglobal] * memory_dvy_dx[offset] + a_x[iglobal] * value_dvy_dx;
				memory_dvx_dy[offset] = b_y_half[jglobal] * memory_dvx_dy[offset] + a_y_half[jglobal] * value_dvx_dy;

				value_dvy_dx = value_dvy_dx / K_x[iglobal] + memory_dvy_dx[offset];
				value_dvx_dy = value_dvx_dy / K_y_half[jglobal] + memory_dvx_dy[offset];

				sigmaxy[offset] = DELTAT_mu * (value_dvy_dx + value_dvx_dy) + sigmaxy[offset];
			}
		}
	}

}

__global__ void kersigmaxzyz(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *cp, float *cs, float *rho, int *DDIMX, int *DDIMY, int *DDIMZ, float *DELTAT, float *memory_dvz_dx, float *memory_dvx_dz, float *memory_dvz_dy, float *memory_dvy_dz, float *a_x, float *a_z, float *a_y_half, float *a_z_half, float *b_x, float *b_y_half, float *b_z_half, float *K_x, float *K_y_half, float *K_z_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vx, float *vy, float *vz, float *sigmaxz, float *sigmayz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMZ[0];
	int left = offset - 1;
	int ztop = offset + DDIMX[0] * DDIMY[0];
	int ytop = offset + DDIMX[0];
	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;


	if ((index_z >= 1) && (index_z <= DDIMZ[0])) {
		//sigmaxz
		if ((index_y >= 1) && (index_y <= DDIMY[0])) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float vs = (cs[left] + cs[ztop]) / 2;
				float rhos = (rho[left] + rho[ztop]) / 2;

				float mu = rhos*vs*vs;

				float DELTAT_mu = DELTAT[0] * mu;

				float value_dvz_dx = (vz[offset] - vz[left]) * ONE_OVER_DELTAX[0];
				float value_dvx_dz = (vx[ztop] - vx[offset]) * ONE_OVER_DELTAZ[0];

				memory_dvz_dx[offset] = b_x[iglobal] * memory_dvz_dx[offset] + a_x[iglobal] * value_dvz_dx;
				memory_dvx_dz[offset] = b_z_half[kglobal] * memory_dvx_dz[offset] + a_z_half[kglobal] * value_dvx_dz;

				value_dvz_dx = value_dvz_dx / K_x[iglobal] + memory_dvz_dx[offset];
				value_dvx_dz = value_dvx_dz / K_z_half[kglobal] + memory_dvx_dz[offset];

				sigmaxz[offset] = DELTAT_mu * (value_dvz_dx + value_dvx_dz) + sigmaxz[offset];
			}
		}

		//sigmayz
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 1) && (index_z <= DDIMX[0])) {
				float vs = (cs[ytop] + cs[ztop]) / 2;
				float rhos = (rho[ytop] + rho[ztop]) / 2;

				float mu = rhos*vs*vs;

				float DELTAT_mu = DELTAT[0] * mu;

				float value_dvz_dy = (vz[ytop] - vz[offset]) * ONE_OVER_DELTAY[0];
				float value_dvy_dz = (vy[ztop] - vy[offset]) * ONE_OVER_DELTAZ[0];

				memory_dvz_dy[offset] = b_y_half[jglobal] * memory_dvz_dy[offset] + a_y_half[jglobal] * value_dvz_dy;
				memory_dvy_dz[offset] = b_z_half[kglobal] * memory_dvy_dz[offset] + a_z_half[kglobal] * value_dvy_dz;

				value_dvz_dy = value_dvz_dy / K_y_half[jglobal] + memory_dvz_dy[offset];
				value_dvy_dz = value_dvy_dz / K_z_half[kglobal] + memory_dvy_dz[offset];

				sigmayz[offset] = DELTAT_mu * (value_dvz_dy + value_dvy_dz) + sigmayz[offset];
			}
		}
	}

}

__global__ void kervxvy(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *rho, int *DDIMX, int *DDIMY, int *DDIMZ, float *DELTAT, float *sigmaxx, float *sigmaxy, float *sigmaxz, float *sigmayy, float *sigmayz, float *memory_dsigmaxx_dx, float *memory_dsigmaxy_dy, float *memory_dsigmaxz_dz, float *memory_dsigmaxy_dx, float *memory_dsigmayy_dy, float *memory_dsigmayz_dz, float *a_x, float *a_y, float *a_z, float *a_x_half, float *a_y_half, float *b_x, float *b_y, float *b_z, float *b_x_half, float *b_y_half, float *K_x, float *K_y, float *K_z, float *K_x_half, float *K_y_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vx, float *vy) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMZ[0];
	int left = offset - 1;
	int ybottom = offset - DDIMX[0];
	int zbottom = offset - DDIMX[0] * DDIMY[0];
	int right = offset + 1;
	int ytop = offset + DDIMX[0];
	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;


	if ((index_z >= 2) && (index_z <= DDIMZ[0])) {
		//vx
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float rhos = (rho[offset] + rho[left]) / 2;

				float DELTAT_over_rho = DELTAT[0] / rhos;

				float value_dsigmaxx_dx = (sigmaxx[offset] - sigmaxx[left]) * ONE_OVER_DELTAX[0];
				float value_dsigmaxy_dy = (sigmaxy[offset] - sigmaxy[ybottom]) * ONE_OVER_DELTAY[0];
				float value_dsigmaxz_dz = (sigmaxz[offset] - sigmaxz[zbottom]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxx_dx[offset] = b_x[iglobal] * memory_dsigmaxx_dx[offset] + a_x[iglobal] * value_dsigmaxx_dx;
				memory_dsigmaxy_dy[offset] = b_y[jglobal] * memory_dsigmaxy_dy[offset] + a_y[jglobal] * value_dsigmaxy_dy;
				memory_dsigmaxz_dz[offset] = b_z[kglobal] * memory_dsigmaxz_dz[offset] + a_z[kglobal] * value_dsigmaxz_dz;

				value_dsigmaxx_dx = value_dsigmaxx_dx / K_x[iglobal] + memory_dsigmaxx_dx[offset];
				value_dsigmaxy_dy = value_dsigmaxy_dy / K_y[jglobal] + memory_dsigmaxy_dy[offset];
				value_dsigmaxz_dz = value_dsigmaxz_dz / K_z[kglobal] + memory_dsigmaxz_dz[offset];

				vx[offset] = DELTAT_over_rho * (value_dsigmaxx_dx + value_dsigmaxy_dy + value_dsigmaxz_dz) + vx[offset];
			}
		}

		//vy
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float rhos = (rho[offset] + rho[ytop]) / 2;

				float DELTAT_over_rho = DELTAT[0] / rhos;

				float value_dsigmaxy_dx = (sigmaxy[right] - sigmaxy[offset]) * ONE_OVER_DELTAX[0];
				float value_dsigmayy_dy = (sigmayy[ytop] - sigmayy[offset]) * ONE_OVER_DELTAY[0];
				float value_dsigmayz_dz = (sigmayz[offset] - sigmayz[zbottom]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxy_dx[offset] = b_x_half[iglobal] * memory_dsigmaxy_dx[offset] + a_x_half[iglobal] * value_dsigmaxy_dx;
				memory_dsigmayy_dy[offset] = b_y_half[jglobal] * memory_dsigmayy_dy[offset] + a_y_half[jglobal] * value_dsigmayy_dy;
				memory_dsigmayz_dz[offset] = b_z[kglobal] * memory_dsigmayz_dz[offset] + a_z[kglobal] * value_dsigmayz_dz;

				value_dsigmaxy_dx = value_dsigmaxy_dx / K_x_half[iglobal] + memory_dsigmaxy_dx[offset];
				value_dsigmayy_dy = value_dsigmayy_dy / K_y_half[jglobal] + memory_dsigmayy_dy[offset];
				value_dsigmayz_dz = value_dsigmayz_dz / K_z[kglobal] + memory_dsigmayz_dz[offset];

				vy[offset] = DELTAT_over_rho * (value_dsigmaxy_dx + value_dsigmayy_dy + value_dsigmayz_dz) + vy[offset];
			}
		}
	}

}

__global__ void kervz(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *rho, int *DDIMX, int *DDIMY, int *DDIMZ, float *DELTAT, float *sigmaxz, float *sigmayz, float *sigmazz, float *memory_dsigmaxz_dx, float *memory_dsigmayz_dy, float *memory_dsigmazz_dz, float *b_x_half, float *b_y, float *b_z_half, float *a_x_half, float *a_y, float *a_z_half, float *K_x_half, float *K_y, float *K_z_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMZ[0];
	int right = offset + 1;
	int ybottom = offset - DDIMX[0];
	int ztop = offset + DDIMX[0] * DDIMY[0];
	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;


	if ((index_z >= 1) && (index_z <= DDIMZ[0] - 1)) {
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float rhos = (rho[offset] + rho[ztop]) / 2;

				float DELTAT_over_rho = DELTAT[0] / rhos;

				float value_dsigmaxz_dx = (sigmaxz[right] - sigmaxz[offset]) * ONE_OVER_DELTAX[0];
				float value_dsigmayz_dy = (sigmayz[offset] - sigmayz[ybottom]) * ONE_OVER_DELTAY[0];
				float value_dsigmazz_dz = (sigmazz[ztop] - sigmazz[offset]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxz_dx[offset] = b_x_half[iglobal] * memory_dsigmaxz_dx[offset] + a_x_half[iglobal] * value_dsigmaxz_dx;
				memory_dsigmayz_dy[offset] = b_y[jglobal] * memory_dsigmayz_dy[offset] + a_y[jglobal] * value_dsigmayz_dy;
				memory_dsigmazz_dz[offset] = b_z_half[kglobal] * memory_dsigmazz_dz[offset] + a_z_half[kglobal] * value_dsigmazz_dz;

				value_dsigmaxz_dx = value_dsigmaxz_dx / K_x_half[iglobal] + memory_dsigmaxz_dx[offset];
				value_dsigmayz_dy = value_dsigmayz_dy / K_y[jglobal] + memory_dsigmayz_dy[offset];
				value_dsigmazz_dz = value_dsigmazz_dz / K_z_half[kglobal] + memory_dsigmazz_dz[offset];

				vz[offset] = DELTAT_over_rho * (value_dsigmaxz_dx + value_dsigmayz_dy + value_dsigmazz_dz) + vz[offset];
			}
		}
	}

}

__global__ void keraddSource(int *ISLBEGIN, int *JSLBEGIN, int *KSLBEGIN, float *sigmaxx, float *sigmayy, float *sigmazz, float *cp, float *cs, float *rho, int *DDIMX, int *DDIMY, int *DDIMZ, int *iit, int *ISOURCE, int *JSOURCE, int *KSOURCE, float *ANGLE_FORCE, float *DEGREES_TO_RADIANS, float *DELTAT, float *factor, float *t0, float *ff0, float *DPI, float *vx, float *vy) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = index_x + index_y*DDIMX[0] + index_z*DDIMX[0] * DDIMY[0];
	int left = offset - 1;
	int ytop = offset + DDIMX[0];

	int iglobal = index_x + ISLBEGIN[0] - 1;
	int jglobal = index_y + JSLBEGIN[0] - 1;
	int kglobal = index_z + KSLBEGIN[0] - 1;

	float lambdaplus2mu = rho[offset] * cp[offset] * cp[offset];

	float a = DPI[0] * DPI[0] * ff0[0] * ff0[0];
	float t = float(iit[0] - 1)*DELTAT[0];

	//Gaussian
	//float source_term = factor * expf(-a*powf((t - t0), 2));

	//first derivative of a Gaussian
	float source_term = -factor[0] * 2.0*a*(t - t0[0])*expf(-a*powf((t - t0[0]), 2));

	//Ricker source time function(second derivative of a Gaussian)
	//float source_term = factor*(1.0 - 2.0*a*powf((t - t0), 2))*expf(-a*powf(t - t0, 2));

	float force_x = sinf(ANGLE_FORCE[0] * DEGREES_TO_RADIANS[0])*source_term;
	float force_y = cosf(ANGLE_FORCE[0] * DEGREES_TO_RADIANS[0])*source_term;

	if (kglobal == KSOURCE[0]) {
		if (jglobal == JSOURCE[0]) {
			if (iglobal == ISOURCE[0]) {
				/*earthquake event source
				vx[offset] = vx[offset] + force_x*DELTAT[0] / ((rho[offset] + rho[left]) / 2);
				vy[offset] = vy[offset] + force_y*DELTAT[0] / ((rho[offset] + rho[ytop]) / 2);
				*/

				/*explosives source*/
				sigmaxx[offset] = sigmaxx[offset] + force_x*DELTAT[0] * lambdaplus2mu;
				sigmayy[offset] = sigmayy[offset] + force_x*DELTAT[0] * lambdaplus2mu;
				sigmazz[offset] = sigmazz[offset] + force_y*DELTAT[0] * lambdaplus2mu;
			}
		}
	}
}

int main(void) {
	int DIMGLOBX = 200;
	int DIMGLOBY = 200;
	int DIMGLOBZ = 100;

	int Ngatx = 10; //jml receiver x
	int Ngaty = 10; //jml receiver y
	int Dgatz = 15; //kedalaman receiver

	int *DDIMGLOBX, *DDIMGLOBY, *DDIMGLOBZ;
	HANDLE_ERROR(hipMalloc((void**)&DDIMGLOBX, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMGLOBY, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMGLOBZ, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(DDIMGLOBX, &DIMGLOBX, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMGLOBY, &DIMGLOBY, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMGLOBZ, &DIMGLOBZ, sizeof(int), hipMemcpyHostToDevice));

	int DIMX = 50;
	int DIMY = 50;
	int DIMZ = 50;

	int offsetperslice = 2; //stagered grid 2nd order space = 2;

	int NSTEP = 1000;
	float DELTATT = 1e-3;
	int sampgat = 2; //tsamp = sampgat*Deltat
	int IT_OUTPUT = 200;

	int DELTAX, DELTAY, DELTAZ;
	DELTAX = 10; DELTAY = DELTAX; DELTAZ = DELTAX;
	float ONE_OVER_DELTAXX, ONE_OVER_DELTAYY, ONE_OVER_DELTAZZ;
	ONE_OVER_DELTAXX = 1 / float(DELTAX);
	ONE_OVER_DELTAZZ = ONE_OVER_DELTAXX; ONE_OVER_DELTAYY = ONE_OVER_DELTAXX;

	float *ONE_OVER_DELTAX, *ONE_OVER_DELTAY, *ONE_OVER_DELTAZ;
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAX, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAY, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAZ, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAX, &ONE_OVER_DELTAXX, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAY, &ONE_OVER_DELTAYY, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAZ, &ONE_OVER_DELTAZZ, sizeof(float), hipMemcpyHostToDevice));

	float *tempcp = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempcs = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *temprho = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	for (int k = 0; k < (DIMGLOBZ + 1); k++) {
		for (int j = 0; j < (DIMGLOBY + 1); j++) {
			for (int i = 0; i < (DIMGLOBX + 1); i++) {
				int ijk = i + j*DIMGLOBX + k*DIMGLOBX*DIMGLOBY;
				tempcp[ijk] = 3300;
				tempcs[ijk] = 3300 / 1.732;
				temprho[ijk] = 3000;
				if (k >= 50) {
					tempcp[ijk] = 2000;
					tempcs[ijk] = 2000 / 1.732;
					temprho[ijk] = 1700;
				}
			}
		}
	}
	float *cp, *cs, *rho;
	HANDLE_ERROR(hipMalloc((void**)&cp, ((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1))*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&cs, ((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1))*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&rho, ((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1))*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(cp, tempcp, sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(cs, tempcs, sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(rho, temprho, sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)), hipMemcpyHostToDevice));
	free(tempcp); free(tempcs); free(temprho);

	float *DELTAT;
	HANDLE_ERROR(hipMalloc((void**)&DELTAT, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DELTAT, &DELTATT, sizeof(float), hipMemcpyHostToDevice));

	float f0, tt0, factorr;
	f0 = 35;
	tt0 = 1.2 / f0;
	float *ff0, *t0;
	HANDLE_ERROR(hipMalloc((void**)&t0, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(t0, &tt0, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&ff0, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ff0, &f0, sizeof(float), hipMemcpyHostToDevice));
	factorr = 1e+7;
	float *factor;
	HANDLE_ERROR(hipMalloc((void**)&factor, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(factor, &factorr, sizeof(float), hipMemcpyHostToDevice));
	int NPOINTS_PML = 10;
	int *DPML;
	HANDLE_ERROR(hipMalloc((void**)&DPML, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(DPML, &NPOINTS_PML, sizeof(int), hipMemcpyHostToDevice));

	int ISOURCEE, KSOURCEE, JSOURCEE;
	ISOURCEE = DIMGLOBX / 2;
	JSOURCEE = DIMGLOBY / 2;
	KSOURCEE = DIMGLOBZ / 2;
	int *ISOURCE, *KSOURCE, *JSOURCE;
	HANDLE_ERROR(hipMalloc((void**)&ISOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(ISOURCE, &ISOURCEE, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&JSOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(JSOURCE, &JSOURCEE, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&KSOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(KSOURCE, &KSOURCEE, sizeof(int), hipMemcpyHostToDevice));

	float ANGLE_FORCEE = 0;
	float *ANGLE_FORCE;
	HANDLE_ERROR(hipMalloc((void**)&ANGLE_FORCE, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ANGLE_FORCE, &ANGLE_FORCEE, sizeof(float), hipMemcpyHostToDevice));

	float PI = 3.141592653589793238462643;
	float *DPI;
	HANDLE_ERROR(hipMalloc((void**)&DPI, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DPI, &PI, sizeof(float), hipMemcpyHostToDevice));
	float DEGREES_TO_RADIANSS = PI / 180;
	float *DEGREES_TO_RADIANS;
	HANDLE_ERROR(hipMalloc((void**)&DEGREES_TO_RADIANS, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DEGREES_TO_RADIANS, &DEGREES_TO_RADIANSS, sizeof(float), hipMemcpyHostToDevice));

	float NPOWER = 2;
	float K_MAX_PML = 1;
	float ALPHA_MAX_PML = 2 * PI*(f0 / 2);

	float *d_x, *K_x, *alpha_x, *a_x, *b_x, *d_x_half, *K_x_half, *alpha_x_half, *a_x_half, *b_x_half;
	HANDLE_ERROR(hipMalloc((void**)&d_x, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_x, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_x, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_x, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_x, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_x_half, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_x_half, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_x_half, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_x_half, (DIMGLOBX + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_x_half, (DIMGLOBX + 1)*sizeof(float)));

	float *d_y, *K_y, *alpha_y, *a_y, *b_y, *d_y_half, *K_y_half, *alpha_y_half, *a_y_half, *b_y_half;
	HANDLE_ERROR(hipMalloc((void**)&d_y, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_y, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_y, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_y, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_y, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_y_half, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_y_half, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_y_half, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_y_half, (DIMGLOBY + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_y_half, (DIMGLOBY + 1)*sizeof(float)));

	float *d_z, *K_z, *alpha_z, *a_z, *b_z, *d_z_half, *K_z_half, *alpha_z_half, *a_z_half, *b_z_half;
	HANDLE_ERROR(hipMalloc((void**)&d_z, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_z, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_z, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_z, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_z, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_z_half, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_z_half, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_z_half, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_z_half, (DIMGLOBZ + 1)*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_z_half, (DIMGLOBZ + 1)*sizeof(float)));

	float thickness_PML_x, thickness_PML_y, thickness_PML_z;
	float xoriginleft, xoriginright, yoriginbottom, yorigintop, zoriginbottom, zorigintop;
	float Rcoef, d0_x, d0_y, d0_z, xval, yval, zval, abscissa_in_PML, abscissa_normalized;

	float Courant_number;

	thickness_PML_x = NPOINTS_PML * DELTAX;
	thickness_PML_y = NPOINTS_PML * DELTAY;
	thickness_PML_z = NPOINTS_PML * DELTAZ;
	Rcoef = 0.001;

	float vpml = 3000;
	d0_x = -(NPOWER + 1) * vpml * logf(Rcoef) / (2.0 * thickness_PML_x);
	d0_y = -(NPOWER + 1) * vpml * logf(Rcoef) / (2.0 * thickness_PML_y);
	d0_z = -(NPOWER + 1) * vpml * logf(Rcoef) / (2.0 * thickness_PML_z);

	//------------------PML X
	float *tempd_x = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempd_x_half = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempa_x = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempa_x_half = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempb_x = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempb_x_half = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempK_x = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempK_x_half = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempalpha_x = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));
	float *tempalpha_x_half = (float*)malloc(sizeof(float)*(DIMGLOBX + 1));

	for (int i = 1; i <= DIMGLOBX; i++) {
		tempd_x[i] = 0.0;
		tempd_x_half[i] = 0.0;
		tempK_x[i] = 1.0;
		tempK_x_half[i] = 1.0;
		tempalpha_x[i] = 0.0;
		tempalpha_x_half[i] = 0.0;
		tempa_x[i] = 0.0;
		tempa_x_half[i] = 0.0;
		tempb_x[i] = 0.0;
		tempb_x_half[i] = 0.0;
	}

	xoriginleft = thickness_PML_x;
	xoriginright = (DIMGLOBX - 1)*DELTAX - thickness_PML_x;
	for (int i = 1; i <= DIMGLOBX; i++) {
		xval = DELTAX*float(i - 1);
		abscissa_in_PML = xoriginleft - xval;//PML XMIN
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xoriginleft - (xval + DELTAX / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x_half[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xval - xoriginright;//PML XMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xval + DELTAX / 2.0 - xoriginright;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x_half[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}
		if (tempalpha_x[i] < 0.0) { tempalpha_x[i] = 0.0; }
		if (tempalpha_x_half[i] < 0.0) { tempalpha_x_half[i] = 0.0; }
		tempb_x[i] = expf(-(tempd_x[i] / tempK_x[i] + tempalpha_x[i])*DELTATT);
		tempb_x_half[i] = expf(-(tempd_x_half[i] / tempK_x_half[i] + tempalpha_x_half[i])*DELTATT);

		if (fabs(tempd_x[i]) > 1e-6) { tempa_x[i] = tempd_x[i] * (tempb_x[i] - 1.0) / (tempK_x[i] * (tempd_x[i] + tempK_x[i] * tempalpha_x[i])); }
		if (fabs(tempd_x_half[i]) > 1e-6) { tempa_x_half[i] = tempd_x_half[i] * (tempb_x_half[i] - 1.0) / (tempK_x_half[i] * (tempd_x_half[i] + tempK_x_half[i] * tempalpha_x_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_x, tempd_x, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_x_half, tempd_x_half, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_x, tempa_x, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_x_half, tempa_x_half, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_x, tempalpha_x, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_x_half, tempalpha_x_half, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_x, tempb_x, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_x_half, tempb_x_half, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_x, tempK_x, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_x_half, tempK_x_half, sizeof(float)*(DIMGLOBX + 1), hipMemcpyHostToDevice));

	//-----------------PML Y
	float *tempd_y = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempd_y_half = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempa_y = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempa_y_half = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempb_y = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempb_y_half = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempK_y = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempK_y_half = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempalpha_y = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));
	float *tempalpha_y_half = (float*)malloc(sizeof(float)*(DIMGLOBY + 1));

	for (int i = 1; i < (DIMGLOBY + 1); i++) {
		tempd_y[i] = 0.0;
		tempd_y_half[i] = 0.0;
		tempK_y[i] = 1.0;
		tempK_y_half[i] = 1.0;
		tempalpha_y[i] = 0.0;
		tempalpha_y_half[i] = 0.0;
		tempa_y[i] = 0.0;
		tempa_y_half[i] = 0.0;
		tempb_y[i] = 0.0;
		tempb_y_half[i] = 0.0;
	}

	yoriginbottom = thickness_PML_y;
	yorigintop = (DIMGLOBY - 1)*DELTAY - thickness_PML_y;
	for (int i = 1; i <= DIMGLOBY; i++) {
		yval = DELTAY*float(i - 1);
		abscissa_in_PML = yoriginbottom - yval;//PML YMIN
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yoriginbottom - (yval + DELTAY / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y_half[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yval - yorigintop;//PML YMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yval + DELTAY / 2.0 - yorigintop;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y_half[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		tempb_y[i] = expf(-(tempd_y[i] / tempK_y[i] + tempalpha_y[i])*DELTATT);
		tempb_y_half[i] = expf(-(tempd_y_half[i] / tempK_y_half[i] + tempalpha_y_half[i])*DELTATT);

		if (fabs(tempd_y[i]) > 1e-6) { tempa_y[i] = tempd_y[i] * (tempb_y[i] - 1.0) / (tempK_y[i] * (tempd_y[i] + tempK_y[i] * tempalpha_y[i])); }
		if (fabs(tempd_y_half[i]) > 1e-6) { tempa_y_half[i] = tempd_y_half[i] * (tempb_y_half[i] - 1.0) / (tempK_y_half[i] * (tempd_y_half[i] + tempK_y_half[i] * tempalpha_y_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_y, tempd_y, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_y_half, tempd_y_half, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_y, tempa_y, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_y_half, tempa_y_half, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_y, tempalpha_y, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_y_half, tempalpha_y_half, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_y, tempb_y, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_y_half, tempb_y_half, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_y, tempK_y, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_y_half, tempK_y_half, sizeof(float)*(DIMGLOBY + 1), hipMemcpyHostToDevice));

	//-----------------PML Z
	float *tempd_z = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempd_z_half = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempa_z = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempa_z_half = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempb_z = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempb_z_half = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempK_z = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempK_z_half = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempalpha_z = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));
	float *tempalpha_z_half = (float*)malloc(sizeof(float)*(DIMGLOBZ + 1));

	for (int i = 1; i < (DIMGLOBZ + 1); i++) {
		tempd_z[i] = 0.0;
		tempd_z_half[i] = 0.0;
		tempK_z[i] = 1.0;
		tempK_z_half[i] = 1.0;
		tempalpha_z[i] = 0.0;
		tempalpha_z_half[i] = 0.0;
		tempa_z[i] = 0.0;
		tempa_z_half[i] = 0.0;
		tempb_z[i] = 0.0;
		tempb_z_half[i] = 0.0;
	}

	zoriginbottom = thickness_PML_z;
	zorigintop = (DIMGLOBZ - 1)*DELTAZ - thickness_PML_z;
	for (int i = 1; i <= DIMGLOBZ; i++) {
		zval = DELTAZ*float(i - 1);
		abscissa_in_PML = zoriginbottom - zval; //PML ZMIN
		// disable pml zmin for free surface condition
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}
		abscissa_in_PML = yoriginbottom - (yval + DELTAY / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z_half[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = zval - zorigintop;//PML ZMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_z;
			tempd_z[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = zval + DELTAZ / 2.0 - zorigintop;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z_half[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		tempb_z[i] = expf(-(tempd_z[i] / tempK_z[i] + tempalpha_z[i])*DELTATT);
		tempb_z_half[i] = expf(-(tempd_z_half[i] / tempK_z_half[i] + tempalpha_z_half[i])*DELTATT);

		if (fabs(tempd_z[i]) > 1e-6) { tempa_z[i] = tempd_z[i] * (tempb_z[i] - 1.0) / (tempK_z[i] * (tempd_z[i] + tempK_z[i] * tempalpha_z[i])); }
		if (fabs(tempd_z_half[i]) > 1e-6) { tempa_z_half[i] = tempd_z_half[i] * (tempb_z_half[i] - 1.0) / (tempK_z_half[i] * (tempd_z_half[i] + tempK_z_half[i] * tempalpha_z_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_z, tempd_z, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_z_half, tempd_z_half, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_z, tempa_z, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_z_half, tempa_z_half, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_z, tempalpha_z, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_z_half, tempalpha_z_half, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_z, tempb_y, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_z_half, tempb_z_half, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_z, tempK_z, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_z_half, tempK_z_half, sizeof(float)*(DIMGLOBZ + 1), hipMemcpyHostToDevice));

	int *DDIMX, *DDIMY, *DDIMZ;
	HANDLE_ERROR(hipMalloc((void**)&DDIMX, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMY, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMZ, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(DDIMX, &DIMX, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMY, &DIMY, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMZ, &DIMZ, sizeof(int), hipMemcpyHostToDevice));

	float *tempvx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempvy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempvz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmaxx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmaxy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmayy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmazz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmaxz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempsigmayz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvx_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvx_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvx_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvy_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvy_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvy_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvz_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvz_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dvz_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmaxx_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmayy_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmazz_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmaxy_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmaxy_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmaxz_dx = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmaxz_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmayz_dy = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	float *tempmemory_dsigmayz_dz = (float*)malloc(sizeof(float)*((DIMGLOBX + 1)*(DIMGLOBY + 1)*(DIMGLOBZ + 1)));
	for (int k = 0; k <= DIMGLOBZ; k++) {
		for (int j = 0; j <= DIMGLOBY; j++) {
			for (int i = 0; i <= DIMGLOBX; i++) {
				int ijk = i + j*DIMGLOBX + k*DIMGLOBX*DIMGLOBY;
				tempvx[ijk] = 0;
				tempvy[ijk] = 0;
				tempvz[ijk] = 0;
				tempsigmaxx[ijk] = 0;
				tempsigmaxy[ijk] = 0;
				tempsigmayy[ijk] = 0;
				tempsigmazz[ijk] = 0;
				tempsigmaxz[ijk] = 0;
				tempsigmayz[ijk] = 0;
				tempmemory_dvx_dx[ijk] = 0;
				tempmemory_dvx_dy[ijk] = 0;
				tempmemory_dvx_dz[ijk] = 0;
				tempmemory_dvy_dx[ijk] = 0;
				tempmemory_dvy_dy[ijk] = 0;
				tempmemory_dvy_dz[ijk] = 0;
				tempmemory_dvz_dx[ijk] = 0;
				tempmemory_dvz_dy[ijk] = 0;
				tempmemory_dvz_dz[ijk] = 0;
				tempmemory_dsigmaxx_dx[ijk] = 0;
				tempmemory_dsigmayy_dy[ijk] = 0;
				tempmemory_dsigmazz_dz[ijk] = 0;
				tempmemory_dsigmaxy_dx[ijk] = 0;
				tempmemory_dsigmaxy_dy[ijk] = 0;
				tempmemory_dsigmaxz_dx[ijk] = 0;
				tempmemory_dsigmaxz_dz[ijk] = 0;
				tempmemory_dsigmayz_dy[ijk] = 0;
				tempmemory_dsigmayz_dz[ijk] = 0;
			}
		}
	}

	for (int it = 1; it <= NSTEP; it++) {
		int *iit;
		HANDLE_ERROR(hipMalloc((void**)&iit, sizeof(int)));
		HANDLE_ERROR(hipMemcpy(iit, &it, sizeof(int), hipMemcpyHostToDevice));
		for (int kk = 2; kk <= DIMGLOBZ; kk += DIMZ) {
			for (int jj = 2; jj <= DIMGLOBY; jj += DIMY) {
				for (int ii = 2; ii <= DIMGLOBX; ii += DIMX) {
					// ukuran per slice ---------------------------------------------------------------------------------------------
					int DLOCALDIMZ = DIMZ + offsetperslice;
					int DLOCALDIMY = DIMY + offsetperslice;
					int DLOCALDIMX = DIMX + offsetperslice;
					if ((kk + DIMZ) > DIMGLOBZ) {
						DLOCALDIMZ = (DIMGLOBZ - kk) + offsetperslice;
					}
					if ((jj + DIMY) > DIMGLOBY) {
						DLOCALDIMY = (DIMGLOBY - jj) + offsetperslice;
					}
					if ((ii + DIMX) > DIMGLOBX) {
						DLOCALDIMX = (DIMGLOBX - ii) + offsetperslice;
					}
					int dd = (DLOCALDIMX + 1)*(DLOCALDIMY + 1)*(DLOCALDIMZ + 1);

					int *DDLOCALDIMX, *DDLOCALDIMY, *DDLOCALDIMZ;
					HANDLE_ERROR(hipMalloc((void**)&DDLOCALDIMX, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&DDLOCALDIMY, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&DDLOCALDIMZ, sizeof(int)));
					HANDLE_ERROR(hipMemcpy(DDLOCALDIMX, &DLOCALDIMX, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DDLOCALDIMY, &DLOCALDIMY, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DDLOCALDIMZ, &DLOCALDIMZ, sizeof(int), hipMemcpyHostToDevice));

					// slicing -------------------------------------------------------------------------------------------------
					int kslbegin = kk - 1;
					int kslend = kk + DIMZ - 1;
					int jslbegin = jj - 1;
					int jslend = jj + DIMY - 1;
					int islbegin = ii - 1;
					int islend = ii + DIMX - 1;
					if ((kk + DIMZ) > DIMGLOBZ) {
						kslbegin = kk - 1;
						kslend = DIMGLOBZ;
					}
					if ((jj + DIMY) > DIMGLOBY) {
						jslbegin = jj - 1;
						jslend = DIMGLOBY;
					}
					if ((ii + DIMX) > DIMGLOBX) {
						islbegin = ii - 1;
						islend = DIMGLOBX;
					}

					int *ISLBEGIN, *JSLBEGIN, *KSLBEGIN;
					HANDLE_ERROR(hipMalloc((void**)&ISLBEGIN, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&JSLBEGIN, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&KSLBEGIN, sizeof(int)));
					HANDLE_ERROR(hipMemcpy(ISLBEGIN, &islbegin, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(JSLBEGIN, &jslbegin, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(KSLBEGIN, &kslbegin, sizeof(int), hipMemcpyHostToDevice));
					int *ISLEND, *JSLEND, *KSLEND;
					HANDLE_ERROR(hipMalloc((void**)&ISLEND, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&JSLEND, sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&KSLEND, sizeof(int)));
					HANDLE_ERROR(hipMemcpy(ISLEND, &islend, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(JSLEND, &jslend, sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(KSLEND, &kslend, sizeof(int), hipMemcpyHostToDevice));

					cout << endl << "kslbegin = " << kslbegin;
					cout << endl << "kslend = " << kslend;
					cout << endl << "DLOCAL DIMZ = " << DLOCALDIMZ;
					cout << endl;
					cout << endl << "jslbegin = " << jslbegin;
					cout << endl << "jslend = " << jslend;
					cout << endl << "DLOCAL DIMY = " << DLOCALDIMY;
					cout << endl;
					cout << endl << "islbegin = " << islbegin;
					cout << endl << "islend = " << islend;
					cout << endl << "DLOCAL DIMX = " << DLOCALDIMX;
					cout << endl;
					cout << "------------------------------------" << endl;
					//getch();

					//alokasi memory ---------------------------------------------------------------------------------------
					float *tempvx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk1 = i + j*DIMX + k*DIMX*DIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvx1[ijk1] = tempvx[ijk2];
							}
						}
					}
					float *vx;
					HANDLE_ERROR(hipMalloc((void**)&vx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(vx, tempvx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempvy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvy1[ijk] = tempvy[ijk2];
							}
						}
					}
					float *vy;
					HANDLE_ERROR(hipMalloc((void**)&vy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(vy, tempvy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempvz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvz1[ijk] = tempvz[ijk2];
							}
						}
					}
					float *vz;
					HANDLE_ERROR(hipMalloc((void**)&vz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(vz, tempvz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmaxx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxx1[ijk] = tempsigmaxx[ijk2];
							}
						}
					}
					float *sigmaxx;
					HANDLE_ERROR(hipMalloc((void**)&sigmaxx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmaxx, tempsigmaxx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmaxy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxy1[ijk] = tempsigmaxy[ijk2];
							}
						}
					}
					float *sigmaxy;
					HANDLE_ERROR(hipMalloc((void**)&sigmaxy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmaxy, tempsigmaxy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmayy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmayy1[ijk] = tempsigmayy[ijk2];
							}
						}
					}
					float *sigmayy;
					HANDLE_ERROR(hipMalloc((void**)&sigmayy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmayy, tempsigmayy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmazz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmazz1[ijk] = tempsigmazz[ijk2];
							}
						}
					}
					float *sigmazz;
					HANDLE_ERROR(hipMalloc((void**)&sigmazz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmazz, tempsigmazz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmaxz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxz1[ijk] = tempsigmaxz[ijk2];
							}
						}
					}
					float *sigmaxz;
					HANDLE_ERROR(hipMalloc((void**)&sigmaxz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmaxz, tempsigmaxz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempsigmayz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmayz1[ijk] = tempsigmayz[ijk2];
							}
						}
					}
					float *sigmayz;
					HANDLE_ERROR(hipMalloc((void**)&sigmayz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(sigmayz, tempsigmayz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvx_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dx1[ijk] = tempmemory_dvx_dx[ijk2];
							}
						}
					}
					float *memory_dvx_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvx_dx, tempmemory_dvx_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvx_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dy1[ijk] = tempmemory_dvx_dy[ijk2];
							}
						}
					}
					float *memory_dvx_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvx_dy, tempmemory_dvx_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvx_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dz1[ijk] = tempmemory_dvx_dz[ijk2];
							}
						}
					}
					float *memory_dvx_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvx_dz, tempmemory_dvx_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvy_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dx1[ijk] = tempmemory_dvy_dx[ijk2];
							}
						}
					}
					float *memory_dvy_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvy_dx, tempmemory_dvy_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvy_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dy1[ijk] = tempmemory_dvy_dy[ijk2];
							}
						}
					}
					float *memory_dvy_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvy_dy, tempmemory_dvy_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvy_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dz1[ijk] = tempmemory_dvy_dz[ijk2];
							}
						}
					}
					float *memory_dvy_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvy_dz, tempmemory_dvy_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvz_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dx1[ijk] = tempmemory_dvz_dx[ijk2];
							}
						}
					}
					float *memory_dvz_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvz_dx, tempmemory_dvz_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvz_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dy1[ijk] = tempmemory_dvz_dy[ijk2];
							}
						}
					}
					float *memory_dvz_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvz_dy, tempmemory_dvz_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dvz_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dz1[ijk] = tempmemory_dvz_dz[ijk2];
							}
						}
					}
					float *memory_dvz_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dvz_dz, tempmemory_dvz_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmaxx_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxx_dx1[ijk] = tempmemory_dsigmaxx_dx[ijk2];
							}
						}
					}
					float *memory_dsigmaxx_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxx_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmaxx_dx, tempmemory_dsigmaxx_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmayy_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayy_dy1[ijk] = tempmemory_dsigmayy_dy[ijk2];
							}
						}
					}
					float *memory_dsigmayy_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayy_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmayy_dy, tempmemory_dsigmayy_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmazz_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmazz_dz1[ijk] = tempmemory_dsigmazz_dz[ijk2];
							}
						}
					}
					float *memory_dsigmazz_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmazz_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmazz_dz, tempmemory_dsigmazz_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmaxy_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxy_dx1[ijk] = tempmemory_dsigmaxy_dx[ijk2];
							}
						}
					}
					float *memory_dsigmaxy_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxy_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmaxy_dx, tempmemory_dsigmaxy_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmaxy_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxy_dy1[ijk] = tempmemory_dsigmaxy_dy[ijk2];
							}
						}
					}
					float *memory_dsigmaxy_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxy_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmaxy_dy, tempmemory_dsigmaxy_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmaxz_dx1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxz_dx1[ijk] = tempmemory_dsigmaxz_dx[ijk2];
							}
						}
					}
					float *memory_dsigmaxz_dx;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxz_dx, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmaxz_dx, tempmemory_dsigmaxz_dx1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmaxz_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxz_dz1[ijk] = tempmemory_dsigmaxz_dz[ijk2];
							}
						}
					}
					float *memory_dsigmaxz_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxz_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmaxz_dz, tempmemory_dsigmaxz_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmayz_dy1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayz_dy1[ijk] = tempmemory_dsigmayz_dy[ijk2];
							}
						}
					}
					float *memory_dsigmayz_dy;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayz_dy, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmayz_dy, tempmemory_dsigmayz_dy1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					float *tempmemory_dsigmayz_dz1 = (float*)malloc(sizeof(float)*dd);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayz_dz1[ijk] = tempmemory_dsigmayz_dz[ijk2];
							}
						}
					}
					float *memory_dsigmayz_dz;
					HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayz_dz, dd*sizeof(float)));
					HANDLE_ERROR(hipMemcpy(memory_dsigmayz_dz, tempmemory_dsigmayz_dz1, sizeof(float)*dd, hipMemcpyHostToDevice));
					
					//run fungsi -----------------------------------------------------------------------------------------------------
					dim3 threads;
					threads.x = 10;
					threads.y = 10;
					threads.z = 10;

					dim3 blocks;
					blocks.x = DLOCALDIMX / threads.x;
					blocks.y = DLOCALDIMY / threads.y;
					blocks.z = DLOCALDIMZ / threads.z;

					kersigmaxyz << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, cp, cs, rho, DELTAT, DDIMX, DDIMY, DDIMZ, memory_dvx_dx, memory_dvy_dy, memory_dvz_dz, a_x_half, a_y, a_z, b_x_half, b_y, b_z, K_x_half, K_y, K_z, sigmaxx, sigmayy, sigmazz, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vx, vy, vz);

					kersigmaxy << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, cp, cs, rho, DDIMX, DDIMY, DDIMZ, DELTAT, memory_dvy_dx, memory_dvx_dy, a_x, a_y_half, b_x, b_y_half, K_x, K_y_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, vx, vy, sigmaxy);

					kersigmaxzyz << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, cp, cs, rho, DDIMX, DDIMY, DDIMZ, DELTAT, memory_dvz_dx, memory_dvx_dz, memory_dvz_dy, memory_dvy_dz, a_x, a_z, a_y_half, a_z_half, b_x, b_y_half, b_z_half, K_x, K_y_half, K_z_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vx, vy, vz, sigmaxz, sigmayz);

					kervxvy << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, rho, DDIMX, DDIMY, DDIMZ, DELTAT, sigmaxx, sigmaxy, sigmaxz, sigmayy, sigmayz, memory_dsigmaxx_dx, memory_dsigmaxy_dy, memory_dsigmaxz_dz, memory_dsigmaxy_dx, memory_dsigmayy_dy, memory_dsigmayz_dz, a_x, a_y, a_z, a_x_half, a_y_half, b_x, b_y, b_z, b_x_half, b_y_half, K_x, K_y, K_z, K_x_half, K_y_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vx, vy);

					kervz << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, rho, DDIMX, DDIMY, DDIMZ, DELTAT, sigmaxz, sigmayz, sigmazz, memory_dsigmaxz_dx, memory_dsigmayz_dy, memory_dsigmazz_dz, b_x_half, b_y, b_z_half, a_x_half, a_y, a_z_half, K_x_half, K_y, K_z_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vz);

					keraddSource << <blocks, threads >> >(ISLBEGIN, JSLBEGIN, KSLBEGIN, sigmaxx, sigmayy, sigmazz, cp, cs, rho, DDIMX, DDIMY, DDIMZ, iit, ISOURCE, JSOURCE, KSOURCE, ANGLE_FORCE, DEGREES_TO_RADIANS, DELTAT, factor, t0, ff0, DPI, vx, vy);

					//copy perslice -> total -----------------------------------------------------------------------------------------
					HANDLE_ERROR(hipMemcpy(tempvx1, vx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(vx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvx[ijk2] = tempvx1[ijk];
							}
						}
					}
					free(tempvx1);

					HANDLE_ERROR(hipMemcpy(tempvy1, vy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(vy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvy[ijk2] = tempvy1[ijk];
							}
						}
					}
					free(tempvy1);

					HANDLE_ERROR(hipMemcpy(tempvz1, vz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(vz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempvz[ijk2] = tempvz1[ijk];
							}
						}
					}
					free(tempvz1);

					HANDLE_ERROR(hipMemcpy(tempsigmaxx1, sigmaxx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmaxx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxx[ijk2] = tempsigmaxx1[ijk];
							}
						}
					}
					free(tempsigmaxx1);

					HANDLE_ERROR(hipMemcpy(tempsigmaxy1, sigmaxy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmaxy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxy[ijk2] = tempsigmaxy1[ijk];
							}
						}
					}
					free(tempsigmaxy1);

					HANDLE_ERROR(hipMemcpy(tempsigmayy1, sigmayy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmayy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmayy[ijk2] = tempsigmayy1[ijk];
							}
						}
					}
					free(tempsigmayy1);

					HANDLE_ERROR(hipMemcpy(tempsigmazz1, sigmazz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmazz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmazz[ijk2] = tempsigmazz1[ijk];
							}
						}
					}
					free(tempsigmazz1);

					HANDLE_ERROR(hipMemcpy(tempsigmaxz1, sigmaxz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmaxz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmaxz[ijk2] = tempsigmaxz1[ijk];
							}
						}
					}
					free(tempsigmaxz1);

					HANDLE_ERROR(hipMemcpy(tempsigmayz1, sigmayz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(sigmayz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempsigmayz[ijk2] = tempsigmayz1[ijk];
							}
						}
					}
					free(tempsigmayz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvx_dx1, memory_dvx_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvx_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dx[ijk2] = tempmemory_dvx_dx1[ijk];
							}
						}
					}
					free(tempmemory_dvx_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvx_dy1, memory_dvx_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvx_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dy[ijk2] = tempmemory_dvx_dy1[ijk];
							}
						}
					}
					free(tempmemory_dvx_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvx_dz1, memory_dvx_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvx_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvx_dz[ijk2] = tempmemory_dvx_dz1[ijk];
							}
						}
					}
					free(tempmemory_dvx_dz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvy_dx1, memory_dvy_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvy_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dx[ijk2] = tempmemory_dvy_dx1[ijk];
							}
						}
					}
					free(tempmemory_dvy_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvy_dy1, memory_dvy_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvy_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dy[ijk2] = tempmemory_dvy_dy1[ijk];
							}
						}
					}
					free(tempmemory_dvy_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvy_dz1, memory_dvy_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvy_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvy_dz[ijk2] = tempmemory_dvy_dz1[ijk];
							}
						}
					}
					free(tempmemory_dvy_dz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvz_dx1, memory_dvz_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvz_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dx[ijk2] = tempmemory_dvz_dx1[ijk];
							}
						}
					}
					free(tempmemory_dvz_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvz_dy1, memory_dvz_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvz_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dy[ijk2] = tempmemory_dvz_dy1[ijk];
							}
						}
					}
					free(tempmemory_dvz_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dvz_dz1, memory_dvz_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dvz_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dvz_dz[ijk2] = tempmemory_dvz_dz1[ijk];
							}
						}
					}
					free(tempmemory_dvz_dz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmaxx_dx1, memory_dsigmaxx_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmaxx_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxx_dx[ijk2] = tempmemory_dsigmaxx_dx1[ijk];
							}
						}
					}
					free(tempmemory_dsigmaxx_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmayy_dy1, memory_dsigmayy_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmayy_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayy_dy[ijk2] = tempmemory_dsigmayy_dy1[ijk];
							}
						}
					}
					free(tempmemory_dsigmayy_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmazz_dz1, memory_dsigmazz_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmazz_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmazz_dz[ijk2] = tempmemory_dsigmazz_dz1[ijk];
							}
						}
					}
					free(tempmemory_dsigmazz_dz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmaxy_dx1, memory_dsigmaxy_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmaxy_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxy_dx[ijk2] = tempmemory_dsigmaxy_dx1[ijk];
							}
						}
					}
					free(tempmemory_dsigmaxy_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmaxy_dy1, memory_dsigmaxy_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmaxy_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxy_dy[ijk2] = tempmemory_dsigmaxy_dy1[ijk];
							}
						}
					}
					free(tempmemory_dsigmaxy_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmaxz_dx1, memory_dsigmaxz_dx, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmaxz_dx);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxz_dx[ijk2] = tempmemory_dsigmaxz_dx1[ijk];
							}
						}
					}
					free(tempmemory_dsigmaxz_dx1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmaxz_dz1, memory_dsigmaxz_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmaxz_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmaxz_dz[ijk2] = tempmemory_dsigmaxz_dz1[ijk];
							}
						}
					}
					free(tempmemory_dsigmaxz_dz1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmayz_dy1, memory_dsigmayz_dy, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmayz_dy);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayz_dy[ijk2] = tempmemory_dsigmayz_dy1[ijk];
							}
						}
					}
					free(tempmemory_dsigmayz_dy1);

					HANDLE_ERROR(hipMemcpy(tempmemory_dsigmayz_dz1, memory_dsigmayz_dz, sizeof(float)*dd, hipMemcpyDeviceToHost));
					hipFree(memory_dsigmayz_dz);
					for (int k = 1; k <= DLOCALDIMZ; k++) {
						for (int j = 1; j <= DLOCALDIMY; j++) {
							for (int i = 1; i <= DLOCALDIMX; i++) {
								int ijk = i + j*DLOCALDIMX + k*DLOCALDIMX*DLOCALDIMY;
								int ijk2 = (i + islbegin - 1) + (j + jslbegin - 1)*DIMGLOBX + (k + kslbegin - 1)*DIMGLOBX*DIMGLOBY;
								tempmemory_dsigmayz_dz[ijk2] = tempmemory_dsigmayz_dz1[ijk];
							}
						}
					}
					free(tempmemory_dsigmayz_dz1);

				}
			}
		}
		//output file gather -----------------------------------------------------------------------------------------------------------------------
		if (fmod(it, sampgat) == 0) {
			char nmfile4[20], nmfile5[20], nmfile6[20];

			int xlen = DIMGLOBX / Ngatx;
			int ylen = (DIMGLOBY- (2 * NPOINTS_PML)) / Ngaty;
			cout << endl << xlen << " " << ylen;
			//sprintf(nmfile4, "rechorvx.bin");
			//std::ofstream fout4(nmfile4, ios::out | ios::app | ios::binary);
			sprintf(nmfile4, "rechorvx.txt");
			std::ofstream fout4;
			fout4.open(nmfile4, ios::app);
			for (int j = 0; j <= DIMGLOBY; j += ylen) {
				for (int i = 0; i <= DIMGLOBX; i += xlen) {
					int kk = i + j*DIMGLOBX + Dgatz*DIMGLOBX*DIMGLOBY;
					fout4 << tempvx[kk] << " ";
				}
			}fout4 << endl;

			//sprintf(nmfile5, "rechorvy.bin");
			//std::ofstream fout5(nmfile5, ios::out | ios::app | ios::binary);
			sprintf(nmfile5, "rechorvy.txt");
			std::ofstream fout5;
			fout5.open(nmfile5, ios::app);
			for (int j = 0; j <= DIMGLOBY; j += ylen) {
				for (int i = 0; i <= DIMGLOBX; i += xlen) {
					int kk = i + j*DIMGLOBX + Dgatz*DIMGLOBX*DIMGLOBY;
					fout5 << tempvy[kk] << " ";
				}
			}fout5 << endl;

			//sprintf(nmfile6, "rechorvz.bin");
			//std::ofstream fout6(nmfile6, ios::out | ios::app | ios::binary);
			sprintf(nmfile6, "rechorvz.txt");
			std::ofstream fout6;
			fout6.open(nmfile6, ios::app);
			for (int j = 0; j <= DIMGLOBY; j += ylen) {
				for (int i = 0; i <= DIMGLOBX; i += xlen) {
					int kk = i + j*DIMGLOBX + Dgatz*DIMGLOBX*DIMGLOBY;
					fout6 << tempvz[kk] << " ";
				}
			}fout6 << endl;
		}

		//output file snap -----------------------------------------------------------------------------------------------------------------------
		if (fmod(it, IT_OUTPUT) == 0){
			//save to file
			char nmfile1[20]; char nmfile2[20]; char nmfile3[20];

			sprintf_s(nmfile1, "vz%05i.bin", it);
			std::ofstream fout1(nmfile1, ios::out | ios::binary);
			//sprintf_s(nmfile1, "vz%05i.txt", it);
			//std::ofstream fout1(nmfile1, ios::out);
			for (int kk = 0; kk < DIMZ; kk++) {
				for (int jj = 0; jj < DIMY; jj++) {
					for (int ii = 0; ii < DIMX; ii++) {
						int ijk = ii + jj*DIMX + kk*DIMX*DIMY;
						fout1.write((char *)&tempvz[ijk], sizeof tempvz[ijk]);
					}
				}
			}

			sprintf_s(nmfile2, "vy%05i.bin", it);
			std::ofstream fout2(nmfile2, ios::out | ios::binary);
			//sprintf_s(nmfile2, "vy%05i.txt", it);
			//std::ofstream fout2(nmfile2, ios::out);
			for (int kk = 0; kk < DIMZ; kk++) {
				for (int jj = 0; jj < DIMY; jj++) {
					for (int ii = 0; ii < DIMX; ii++) {
						int ijk = ii + jj*DIMX + kk*DIMX*DIMY;
						fout2.write((char *)&tempvy[ijk], sizeof tempvy[ijk]);
					}
				}
			}

			sprintf_s(nmfile3, "vx%05i.bin", it);
			std::ofstream fout3(nmfile3, ios::out | ios::binary);
			//sprintf_s(nmfile3, "vx%05i.txt", it);
			//std::ofstream fout3(nmfile3, ios::out);
			for (int kk = 0; kk < DIMZ; kk++) {
				for (int jj = 0; jj < DIMY; jj++) {
					for (int ii = 0; ii < DIMX; ii++) {
						int ijk = ii + jj*DIMX + kk*DIMX*DIMY;
						fout3.write((char *)&tempvx[ijk], sizeof tempvx[ijk]);
					}
				}
			}

			//save to file END
		}
	}
	return 0;
}

#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <stdio.h>
#include <fstream>

static void HandleError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void kersigmaxyz(int *DDIMX, int *DDIMY, int *DDIMZ, float *memory_dvx_dx, float *memory_dvy_dy, float *memory_dvz_dz, float *a_x_half, float *a_y, float *a_z, float *b_x_half, float *b_y, float *b_z, float *K_x_half, float *K_y, float *K_z, float *DELTAT_lambdaplus2mu, float *DELTAT_lambda, float *sigmaxx, float *sigmayy, float *sigmazz, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vx, float *vy, float *vz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;
	int right = offset + 1;
	int ybottom = offset - DDIMX[0];
	int zbottom = offset - DDIMX[0] * DDIMY[0];


	if ((index_z >= 2) && (index_z <= DDIMZ[0])) {
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float value_dvx_dx = (vx[right] - vx[offset])*ONE_OVER_DELTAX[0];
				float value_dvy_dy = (vy[offset] - vy[ybottom])*ONE_OVER_DELTAY[0];
				float value_dvz_dz = (vz[offset] - vz[zbottom])*ONE_OVER_DELTAZ[0];

				memory_dvx_dx[offset] = b_x_half[index_x] * memory_dvx_dx[offset] + a_x_half[index_x] * value_dvx_dx;
				memory_dvy_dy[offset] = b_y[index_y] * memory_dvy_dy[offset] + a_y[index_y] * value_dvy_dy;
				memory_dvz_dz[offset] = b_z[index_z] * memory_dvz_dz[offset] + a_z[index_z] * value_dvz_dz;

				value_dvx_dx = value_dvx_dx / K_x_half[index_x] + memory_dvx_dx[offset];
				value_dvy_dy = value_dvy_dy / K_y[index_y] + memory_dvy_dy[offset];
				value_dvz_dz = value_dvz_dz / K_z[index_z] + memory_dvz_dz[offset];

				sigmaxx[offset] = DELTAT_lambdaplus2mu[0] * value_dvx_dx + DELTAT_lambda[0] * (value_dvy_dy + value_dvz_dz) + sigmaxx[offset];
				sigmayy[offset] = DELTAT_lambda[0] * (value_dvx_dx + value_dvz_dz) + DELTAT_lambdaplus2mu[0] * value_dvy_dy + sigmayy[offset];
				sigmazz[offset] = DELTAT_lambda[0] * (value_dvx_dx + value_dvy_dy) + DELTAT_lambdaplus2mu[0] * value_dvz_dz + sigmazz[offset];
			}
		}
	}

}

__global__ void kersigmaxy(int *DDIMX, int *DDIMY, int *DDIMZ, float *memory_dvy_dx, float *memory_dvx_dy, float *a_x, float *a_y_half, float *b_x, float *b_y_half, float *K_x, float *K_y_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *vx, float *vy, float *DELTAT_mu, float *sigmaxy) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;
	int left = offset - 1;
	int ytop = offset + DDIMX[0];


	if ((index_z >= 1) && (index_z <= DDIMZ[0])) {
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float value_dvy_dx = (vy[offset] - vy[left])*ONE_OVER_DELTAX[0];
				float value_dvx_dy = (vx[ytop] - vx[offset])*ONE_OVER_DELTAY[0];

				memory_dvy_dx[offset] = b_x[index_x] * memory_dvy_dx[offset] + a_x[index_x] * value_dvy_dx;
				memory_dvx_dy[offset] = b_y_half[index_y] * memory_dvx_dy[offset] + a_y_half[index_y] * value_dvx_dy;

				value_dvy_dx = value_dvy_dx / K_x[index_x] + memory_dvy_dx[offset];
				value_dvx_dy = value_dvx_dy / K_y_half[index_y] + memory_dvx_dy[offset];

				sigmaxy[offset] = DELTAT_mu[0] * (value_dvy_dx + value_dvx_dy) + sigmaxy[offset];
			}
		}
	}

}

__global__ void kersigmaxzyz(int *DDIMX, int *DDIMY, int *DDIMZ, float *memory_dvz_dx, float *memory_dvx_dz, float *memory_dvz_dy, float *memory_dvy_dz, float *a_x, float *a_z, float *a_y_half, float *a_z_half, float *b_x, float *b_y_half, float *b_z_half, float *K_x, float *K_y_half, float *K_z_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vx, float *vy, float *vz, float *DELTAT_mu, float *sigmaxz, float *sigmayz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;
	int left = offset - 1;
	int ztop = offset + DDIMX[0] * DDIMY[0];
	int ytop = offset + DDIMX[0];


	if ((index_z >= 1) && (index_z <= DDIMZ[0])) {
		//sigmaxz
		if ((index_y >= 1) && (index_y <= DDIMY[0])) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float value_dvz_dx = (vz[offset] - vz[left]) * ONE_OVER_DELTAX[0];
				float value_dvx_dz = (vx[ztop] - vx[offset]) * ONE_OVER_DELTAZ[0];

				memory_dvz_dx[offset] = b_x[index_x] * memory_dvz_dx[offset] + a_x[index_x] * value_dvz_dx;
				memory_dvx_dz[offset] = b_z_half[index_z] * memory_dvx_dz[offset] + a_z_half[index_z] * value_dvx_dz;

				value_dvz_dx = value_dvz_dx / K_x[index_x] + memory_dvz_dx[offset];
				value_dvx_dz = value_dvx_dz / K_z_half[index_z] + memory_dvx_dz[offset];

				sigmaxz[offset] = DELTAT_mu[0] * (value_dvz_dx + value_dvx_dz) + sigmaxz[offset];
			}
		}

		//sigmayz
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 1) && (index_z <= DDIMX[0])) {
				float value_dvz_dy = (vz[ytop] - vz[offset]) * ONE_OVER_DELTAY[0];
				float value_dvy_dz = (vy[ztop] - vy[offset]) * ONE_OVER_DELTAZ[0];

				memory_dvz_dy[offset] = b_y_half[index_y] * memory_dvz_dy[offset] + a_y_half[index_y] * value_dvz_dy;
				memory_dvy_dz[offset] = b_z_half[index_z] * memory_dvy_dz[offset] + a_z_half[index_z] * value_dvy_dz;

				value_dvz_dy = value_dvz_dy / K_y_half[index_y] + memory_dvz_dy[offset];
				value_dvy_dz = value_dvy_dz / K_z_half[index_z] + memory_dvy_dz[offset];

				sigmayz[offset] = DELTAT_mu[0] * (value_dvz_dy + value_dvy_dz) + sigmayz[offset];
			}
		}
	}

}

__global__ void kervxvy(int *DDIMX, int *DDIMY, int *DDIMZ,float *sigmaxx, float *sigmaxy, float *sigmaxz, float *sigmayy, float *sigmayz, float *memory_dsigmaxx_dx, float *memory_dsigmaxy_dy, float *memory_dsigmaxz_dz, float *memory_dsigmaxy_dx, float *memory_dsigmayy_dy, float *memory_dsigmayz_dz, float *a_x, float *a_y, float *a_z, float *a_x_half, float *a_y_half, float *b_x, float *b_y, float *b_z, float *b_x_half, float *b_y_half, float *K_x, float *K_y, float *K_z, float *K_x_half, float *K_y_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *DELTAT_over_rho, float *vx, float *vy) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;
	int left = offset - 1;
	int ybottom = offset - DDIMX[0];
	int zbottom = offset - DDIMX[0] * DDIMY[0];
	int right = offset + 1;
	int ytop = offset + DDIMX[0];


	if ((index_z >= 2) && (index_z <= DDIMZ[0])) {
		//vx
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 2) && (index_z <= DDIMX[0])) {
				float value_dsigmaxx_dx = (sigmaxx[offset] - sigmaxx[left]) * ONE_OVER_DELTAX[0];
				float value_dsigmaxy_dy = (sigmaxy[offset] - sigmaxy[ybottom]) * ONE_OVER_DELTAY[0];
				float value_dsigmaxz_dz = (sigmaxz[offset] - sigmaxz[zbottom]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxx_dx[offset] = b_x[index_x] * memory_dsigmaxx_dx[offset] + a_x[index_x] * value_dsigmaxx_dx;
				memory_dsigmaxy_dy[offset] = b_y[index_y] * memory_dsigmaxy_dy[offset] + a_y[index_y] * value_dsigmaxy_dy;
				memory_dsigmaxz_dz[offset] = b_z[index_z] * memory_dsigmaxz_dz[offset] + a_z[index_z] * value_dsigmaxz_dz;

				value_dsigmaxx_dx = value_dsigmaxx_dx / K_x[index_x] + memory_dsigmaxx_dx[offset];
				value_dsigmaxy_dy = value_dsigmaxy_dy / K_y[index_y] + memory_dsigmaxy_dy[offset];
				value_dsigmaxz_dz = value_dsigmaxz_dz / K_z[index_z] + memory_dsigmaxz_dz[offset];

				vx[offset] = DELTAT_over_rho[0] * (value_dsigmaxx_dx + value_dsigmaxy_dy + value_dsigmaxz_dz) + vx[offset];
			}
		}

		//vy
		if ((index_y >= 1) && (index_y <= DDIMY[0] - 1)) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float value_dsigmaxy_dx = (sigmaxy[right] - sigmaxy[offset]) * ONE_OVER_DELTAX[0];
				float value_dsigmayy_dy = (sigmayy[ytop] - sigmayy[offset]) * ONE_OVER_DELTAY[0];
				float value_dsigmayz_dz = (sigmayz[offset] - sigmayz[zbottom]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxy_dx[offset] = b_x_half[index_x] * memory_dsigmaxy_dx[offset] + a_x_half[index_x] * value_dsigmaxy_dx;
				memory_dsigmayy_dy[offset] = b_y_half[index_y] * memory_dsigmayy_dy[offset] + a_y_half[index_y] * value_dsigmayy_dy;
				memory_dsigmayz_dz[offset] = b_z[index_z] * memory_dsigmayz_dz[offset] + a_z[index_z] * value_dsigmayz_dz;

				value_dsigmaxy_dx = value_dsigmaxy_dx / K_x_half[index_x] + memory_dsigmaxy_dx[offset];
				value_dsigmayy_dy = value_dsigmayy_dy / K_y_half[index_y] + memory_dsigmayy_dy[offset];
				value_dsigmayz_dz = value_dsigmayz_dz / K_z[index_z] + memory_dsigmayz_dz[offset];

				vy[offset] = DELTAT_over_rho[0] * (value_dsigmaxy_dx + value_dsigmayy_dy + value_dsigmayz_dz) + vy[offset];
			}
		}
	}

}

__global__ void kervz(int *DDIMX, int *DDIMY, int *DDIMZ, float *sigmaxz, float *sigmayz, float *sigmazz, float *memory_dsigmaxz_dx, float *memory_dsigmayz_dy, float *memory_dsigmazz_dz, float *b_x_half, float *b_y, float *b_z_half, float *a_x_half, float *a_y, float *a_z_half, float *K_x_half, float *K_y, float *K_z_half, float *ONE_OVER_DELTAX, float *ONE_OVER_DELTAY, float *ONE_OVER_DELTAZ, float *vz, float *DELTAT_over_rho) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;
	int right = offset + 1;
	int ybottom = offset - DDIMX[0];
	int ztop = offset + DDIMX[0] * DDIMY[0];


	if ((index_z >= 1) && (index_z <= DDIMZ[0] - 1)) {
		if ((index_y >= 2) && (index_y <= DDIMY[0])) {
			if ((index_x >= 1) && (index_z <= DDIMX[0] - 1)) {
				float value_dsigmaxz_dx = (sigmaxz[right] - sigmaxz[offset]) * ONE_OVER_DELTAX[0];
				float value_dsigmayz_dy = (sigmayz[offset] - sigmayz[ybottom]) * ONE_OVER_DELTAY[0];
				float value_dsigmazz_dz = (sigmazz[ztop] - sigmazz[offset]) * ONE_OVER_DELTAZ[0];

				memory_dsigmaxz_dx[offset] = b_x_half[index_x] * memory_dsigmaxz_dx[offset] + a_x_half[index_x] * value_dsigmaxz_dx;
				memory_dsigmayz_dy[offset] = b_y[index_y] * memory_dsigmayz_dy[offset] + a_y[index_y] * value_dsigmayz_dy;
				memory_dsigmazz_dz[offset] = b_z_half[index_z] * memory_dsigmazz_dz[offset] + a_z_half[index_z] * value_dsigmazz_dz;

				value_dsigmaxz_dx = value_dsigmaxz_dx / K_x_half[index_x] + memory_dsigmaxz_dx[offset];
				value_dsigmayz_dy = value_dsigmayz_dy / K_y[index_y] + memory_dsigmayz_dy[offset];
				value_dsigmazz_dz = value_dsigmazz_dz / K_z_half[index_z] + memory_dsigmazz_dz[offset];

				vz[offset] = DELTAT_over_rho[0] * (value_dsigmaxz_dx + value_dsigmayz_dy + value_dsigmazz_dz) + vz[offset];
			}
		}
	}

}

__global__ void keraddSource(int *iit, int *ISOURCE, int *JSOURCE, int *KSOURCE, float *ANGLE_FORCE, float *DEGREES_TO_RADIANS, float *DELTAT, float *factor, float *t0, float *ff0, float *DPI, float *vx, float *vy, float *rho) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;

	float a = DPI[0] * DPI[0] * ff0[0] * ff0[0];
	float t = float(iit[0] - 1)*DELTAT[0];

	//Gaussian
	//float source_term = factor * expf(-a*powf((t - t0), 2));

	//first derivative of a Gaussian
	float source_term = -factor[0] * 2.0*a*(t - t0[0])*expf(-a*powf((t - t0[0]), 2));

	//Ricker source time function(second derivative of a Gaussian)
	//float source_term = factor*(1.0 - 2.0*a*powf((t - t0), 2))*expf(-a*powf(t - t0, 2));

	float force_x = sinf(ANGLE_FORCE[0] * DEGREES_TO_RADIANS[0])*source_term;
	float force_y = cosf(ANGLE_FORCE[0] * DEGREES_TO_RADIANS[0])*source_term;

	if (index_z == KSOURCE[0]) {
		if (index_y == JSOURCE[0]) {
			if (index_x == ISOURCE[0]) {
				vx[offset] = vx[offset] + force_x*DELTAT[0] / rho[0];
				vy[offset] = vy[offset] + force_y*DELTAT[0] / rho[0];
			}
		}
	}
}

__global__ void kerDirichletBoundary(int *DDIMX, int *DDIMY, int *DDIMZ, float *vx, float *vy, float *vz) {
	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int index_z = blockIdx.z * blockDim.z + threadIdx.z;

	int blkId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int aaaa = blockDim.x * blockDim.y * blockDim.z;
	int bbbb = threadIdx.z * (blockDim.x * blockDim.y);
	int cccc = (threadIdx.y * blockDim.x) + threadIdx.x;
	int offset = blkId * aaaa + bbbb + cccc;

	if ((index_z == 1) || (index_z == DDIMZ[0])) {
		if ((index_y == 1) || (index_y == DDIMY[0])) {
			if ((index_x == 1) || (index_z == DDIMX[0])) {
				vy[offset] = 0.0;
				vx[offset] = 0.0;
				vz[offset] = 0.0;
			}
		}
	}
}

int main()
{
	int NIMX, NIMY, NIMZ;
	NIMX = 200;
	NIMY = 200;
	NIMZ = 200;

	int NSTEP = 2500;

	int DIMX, DIMY, DIMZ;
	DIMX = NIMX + 1; DIMY = NIMY + 1; DIMZ = NIMZ + 1;

	int DELTAX, DELTAY, DELTAZ;
	DELTAX = 1.5; DELTAY = DELTAX; DELTAZ = DELTAX;
	float ONE_OVER_DELTAXX, ONE_OVER_DELTAYY, ONE_OVER_DELTAZZ;
	ONE_OVER_DELTAXX = 1 / float(DELTAX);
	ONE_OVER_DELTAZZ = ONE_OVER_DELTAXX; ONE_OVER_DELTAYY = ONE_OVER_DELTAXX;

	float *ONE_OVER_DELTAX, *ONE_OVER_DELTAY, *ONE_OVER_DELTAZ;
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAX, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAY, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&ONE_OVER_DELTAZ, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAX, &ONE_OVER_DELTAXX, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAY, &ONE_OVER_DELTAYY, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(ONE_OVER_DELTAZ, &ONE_OVER_DELTAZZ, sizeof(float), hipMemcpyHostToDevice));

	float tempcp, tempcs, temprho, mu, lambda, lambdaplustwomu;
	tempcp = 3300.0;
	tempcs = tempcp / 1.732;
	temprho = 3000.0;
	float *rho;
	HANDLE_ERROR(hipMalloc((void**)&rho, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(rho, &temprho, sizeof(float), hipMemcpyHostToDevice));
	mu = temprho*tempcs*tempcs;
	lambda = temprho*(tempcp*tempcp - 2 * tempcs*tempcs);
	lambdaplustwomu = temprho*tempcp*tempcp;

	float DELTATT = 1e-5;
	float *DELTAT;
	HANDLE_ERROR(hipMalloc((void**)&DELTAT, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DELTAT, &DELTATT, sizeof(float), hipMemcpyHostToDevice));

	float f0, tt0, factorr;
	f0 = 7;
	tt0 = 1.2 / f0;
	float *ff0, *t0;
	HANDLE_ERROR(hipMalloc((void**)&t0, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(t0, &tt0, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&ff0, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ff0, &f0, sizeof(float), hipMemcpyHostToDevice));
	factorr = 1e+7;
	float *factor;
	HANDLE_ERROR(hipMalloc((void**)&factor, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(factor, &factorr, sizeof(float), hipMemcpyHostToDevice));

	int NPOINTS_PML = 10;

	int ISOURCEE, KSOURCEE, JSOURCEE;
	ISOURCEE = (NIMX) / 2;
	JSOURCEE = (NIMY) / 2;
	KSOURCEE = (NIMZ) / 2;
	int *ISOURCE, *KSOURCE, *JSOURCE;
	HANDLE_ERROR(hipMalloc((void**)&ISOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(ISOURCE, &ISOURCEE, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&JSOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(JSOURCE, &JSOURCEE, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMalloc((void**)&KSOURCE, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(KSOURCE, &KSOURCEE, sizeof(int), hipMemcpyHostToDevice));

	float ANGLE_FORCEE = 90;
	float *ANGLE_FORCE;
	HANDLE_ERROR(hipMalloc((void**)&ANGLE_FORCE, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(ANGLE_FORCE, &ANGLE_FORCEE, sizeof(float), hipMemcpyHostToDevice));
	int IT_OUTPUT = 200;

	float PI = 3.141592653589793238462643;
	float *DPI;
	HANDLE_ERROR(hipMalloc((void**)&DPI, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DPI, &PI, sizeof(float), hipMemcpyHostToDevice));
	float DEGREES_TO_RADIANSS = PI / 180;
	float *DEGREES_TO_RADIANS;
	HANDLE_ERROR(hipMalloc((void**)&DEGREES_TO_RADIANS, sizeof(float)));
	HANDLE_ERROR(hipMemcpy(DEGREES_TO_RADIANS, &DEGREES_TO_RADIANSS, sizeof(float), hipMemcpyHostToDevice));

	float NPOWER = 2;
	float K_MAX_PML = 1;
	float ALPHA_MAX_PML = 2 * PI*(f0 / 2);

	float *tempvx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempvx[ijk] = 0;
			}
		}
	}
	float *vx;
	HANDLE_ERROR(hipMalloc((void**)&vx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(vx, tempvx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempvx);

	float *tempvy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempvy[ijk] = 0;
			}
		}
	}
	float *vy;
	HANDLE_ERROR(hipMalloc((void**)&vy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(vy, tempvy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempvy);

	float *tempvz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempvz[ijk] = 0;
			}
		}
	}
	float *vz;
	HANDLE_ERROR(hipMalloc((void**)&vz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(vz, tempvz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempvz);

	float *tempsigmaxx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmaxx[ijk] = 0;
			}
		}
	}
	float *sigmaxx;
	HANDLE_ERROR(hipMalloc((void**)&sigmaxx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmaxx, tempsigmaxx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmaxx);

	float *tempsigmaxy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmaxy[ijk] = 0;
			}
		}
	}
	float *sigmaxy;
	HANDLE_ERROR(hipMalloc((void**)&sigmaxy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmaxy, tempsigmaxy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmaxy);

	float *tempsigmayy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmayy[ijk] = 0;
			}
		}
	}
	float *sigmayy;
	HANDLE_ERROR(hipMalloc((void**)&sigmayy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmayy, tempsigmayy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmayy);

	float *tempsigmazz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmazz[ijk] = 0;
			}
		}
	}
	float *sigmazz;
	HANDLE_ERROR(hipMalloc((void**)&sigmazz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmazz, tempsigmazz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmazz);

	float *tempsigmaxz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmaxz[ijk] = 0;
			}
		}
	}
	float *sigmaxz;
	HANDLE_ERROR(hipMalloc((void**)&sigmaxz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmaxz, tempsigmaxz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmaxz);

	float *tempsigmayz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempsigmayz[ijk] = 0;
			}
		}
	}
	float *sigmayz;
	HANDLE_ERROR(hipMalloc((void**)&sigmayz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(sigmayz, tempsigmayz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempsigmayz);

	float *tempmemory_dvx_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvx_dx[ijk] = 0;
			}
		}
	}
	float *memory_dvx_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvx_dx, tempmemory_dvx_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvx_dx);

	float *tempmemory_dvx_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvx_dy[ijk] = 0;
			}
		}
	}
	float *memory_dvx_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvx_dy, tempmemory_dvx_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvx_dy);

	float *tempmemory_dvx_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvx_dz[ijk] = 0;
			}
		}
	}
	float *memory_dvx_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvx_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvx_dz, tempmemory_dvx_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvx_dz);

	float *tempmemory_dvy_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvy_dx[ijk] = 0;
			}
		}
	}
	float *memory_dvy_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvy_dx, tempmemory_dvy_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvy_dx);

	float *tempmemory_dvy_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvy_dy[ijk] = 0;
			}
		}
	}
	float *memory_dvy_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvy_dy, tempmemory_dvy_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvy_dy);

	float *tempmemory_dvy_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvy_dz[ijk] = 0;
			}
		}
	}
	float *memory_dvy_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvy_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvy_dz, tempmemory_dvy_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvy_dz);

	float *tempmemory_dvz_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvz_dx[ijk] = 0;
			}
		}
	}
	float *memory_dvz_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvz_dx, tempmemory_dvz_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvz_dx);

	float *tempmemory_dvz_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvz_dy[ijk] = 0;
			}
		}
	}
	float *memory_dvz_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvz_dy, tempmemory_dvz_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvz_dy);

	float *tempmemory_dvz_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dvz_dz[ijk] = 0;
			}
		}
	}
	float *memory_dvz_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dvz_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dvz_dz, tempmemory_dvz_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dvz_dz);

	float *tempmemory_dsigmaxx_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmaxx_dx[ijk] = 0;
			}
		}
	}
	float *memory_dsigmaxx_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxx_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmaxx_dx, tempmemory_dsigmaxx_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmaxx_dx);

	float *tempmemory_dsigmayy_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmayy_dy[ijk] = 0;
			}
		}
	}
	float *memory_dsigmayy_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayy_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmayy_dy, tempmemory_dsigmayy_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmayy_dy);

	float *tempmemory_dsigmazz_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmazz_dz[ijk] = 0;
			}
		}
	}
	float *memory_dsigmazz_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmazz_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmazz_dz, tempmemory_dsigmazz_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmazz_dz);

	float *tempmemory_dsigmaxy_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmaxy_dx[ijk] = 0;
			}
		}
	}
	float *memory_dsigmaxy_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxy_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmaxy_dx, tempmemory_dsigmaxy_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmaxy_dx);

	float *tempmemory_dsigmaxy_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmaxy_dy[ijk] = 0;
			}
		}
	}
	float *memory_dsigmaxy_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxy_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmaxy_dy, tempmemory_dsigmaxy_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmaxy_dy);

	float *tempmemory_dsigmaxz_dx = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmaxz_dx[ijk] = 0;
			}
		}
	}
	float *memory_dsigmaxz_dx;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxz_dx, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmaxz_dx, tempmemory_dsigmaxz_dx, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmaxz_dx);

	float *tempmemory_dsigmaxz_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmaxz_dz[ijk] = 0;
			}
		}
	}
	float *memory_dsigmaxz_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmaxz_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmaxz_dz, tempmemory_dsigmaxz_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmaxz_dz);

	float *tempmemory_dsigmayz_dy = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmayz_dy[ijk] = 0;
			}
		}
	}
	float *memory_dsigmayz_dy;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayz_dy, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmayz_dy, tempmemory_dsigmayz_dy, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmayz_dy);

	float *tempmemory_dsigmayz_dz = (float*)malloc(sizeof(float)*(DIMX*DIMY*DIMZ));
	for (int k = 0; k < DIMZ; k++) {
		for (int j = 0; j < DIMY; j++) {
			for (int i = 0; i < DIMX; i++) {
				int ijk = i + j*DIMX + k*DIMX*DIMY;
				tempmemory_dsigmayz_dz[ijk] = 0;
			}
		}
	}
	float *memory_dsigmayz_dz;
	HANDLE_ERROR(hipMalloc((void**)&memory_dsigmayz_dz, DIMX*DIMY*DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMemcpy(memory_dsigmayz_dz, tempmemory_dsigmayz_dz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice));
	free(tempmemory_dsigmayz_dz);

	float *d_x, *K_x, *alpha_x, *a_x, *b_x, *d_x_half, *K_x_half, *alpha_x_half, *a_x_half, *b_x_half;
	HANDLE_ERROR(hipMalloc((void**)&d_x, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_x, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_x, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_x, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_x, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_x_half, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_x_half, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_x_half, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_x_half, DIMX*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_x_half, DIMX*sizeof(float)));

	float *d_y, *K_y, *alpha_y, *a_y, *b_y, *d_y_half, *K_y_half, *alpha_y_half, *a_y_half, *b_y_half;
	HANDLE_ERROR(hipMalloc((void**)&d_y, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_y, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_y, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_y, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_y, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_y_half, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_y_half, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_y_half, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_y_half, DIMY*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_y_half, DIMY*sizeof(float)));

	float *d_z, *K_z, *alpha_z, *a_z, *b_z, *d_z_half, *K_z_half, *alpha_z_half, *a_z_half, *b_z_half;
	HANDLE_ERROR(hipMalloc((void**)&d_z, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_z, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_z, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_z, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_z, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&d_z_half, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&K_z_half, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&alpha_z_half, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&a_z_half, DIMZ*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&b_z_half, DIMZ*sizeof(float)));

	float thickness_PML_x, thickness_PML_y, thickness_PML_z;
	float xoriginleft, xoriginright, yoriginbottom, yorigintop, zoriginbottom, zorigintop;
	float Rcoef, d0_x, d0_y, d0_z, xval, yval, zval, abscissa_in_PML, abscissa_normalized;
	float a, t, force_x, force_y, source_term;
	float epsilon_xx, epsilon_yy, epsilon_zz, epsilon_xy, epsilon_xz, epsilon_yz;
	float total_energy_kinetic, total_energy_potential;
	float *total_energy = (float*)malloc(sizeof(float)*NSTEP);

	float tDELTAT_lambda = DELTATT*lambda;
	float tDELTAT_mu = DELTATT*mu;
	float tDELTAT_lambdaplus2mu = DELTATT*lambdaplustwomu;
	float tDELTAT_over_rho = DELTATT / temprho;
	float Courant_number;

	float *DELTAT_lambda, *DELTAT_mu, *DELTAT_lambdaplus2mu, *DELTAT_over_rho;
	HANDLE_ERROR(hipMalloc((void**)&DELTAT_lambda, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&DELTAT_mu, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&DELTAT_lambdaplus2mu, sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&DELTAT_over_rho, sizeof(float)));

	HANDLE_ERROR(hipMemcpy(DELTAT_lambda, &tDELTAT_lambda, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DELTAT_mu, &tDELTAT_mu, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DELTAT_lambdaplus2mu, &tDELTAT_lambdaplus2mu, sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DELTAT_over_rho, &tDELTAT_over_rho, sizeof(float), hipMemcpyHostToDevice));

	thickness_PML_x = NPOINTS_PML * DELTAX;
	thickness_PML_y = NPOINTS_PML * DELTAY;
	thickness_PML_z = NPOINTS_PML * DELTAZ;
	Rcoef = 0.001;

	d0_x = -(NPOWER + 1) * tempcp * logf(Rcoef) / (2.0 * thickness_PML_x);
	d0_y = -(NPOWER + 1) * tempcp * logf(Rcoef) / (2.0 * thickness_PML_y);
	d0_z = -(NPOWER + 1) * tempcp * logf(Rcoef) / (2.0 * thickness_PML_z);

	//------------------PML X
	float *tempd_x = (float*)malloc(sizeof(float)*DIMX);
	float *tempd_x_half = (float*)malloc(sizeof(float)*DIMX);
	float *tempa_x = (float*)malloc(sizeof(float)*DIMX);
	float *tempa_x_half = (float*)malloc(sizeof(float)*DIMX);
	float *tempb_x = (float*)malloc(sizeof(float)*DIMX);
	float *tempb_x_half = (float*)malloc(sizeof(float)*DIMX);
	float *tempK_x = (float*)malloc(sizeof(float)*DIMX);
	float *tempK_x_half = (float*)malloc(sizeof(float)*DIMX);
	float *tempalpha_x = (float*)malloc(sizeof(float)*DIMX);
	float *tempalpha_x_half = (float*)malloc(sizeof(float)*DIMX);

	for (int i = 1; i < DIMX; i++) {
		tempd_x[i] = 0.0;
		tempd_x_half[i] = 0.0;
		tempK_x[i] = 1.0;
		tempK_x_half[i] = 1.0;
		tempalpha_x[i] = 0.0;
		tempalpha_x_half[i] = 0.0;
		tempa_x[i] = 0.0;
		tempa_x_half[i] = 0.0;
		tempb_x[i] = 0.0;
		tempb_x_half[i] = 0.0;
	}

	xoriginleft = thickness_PML_x;
	xoriginright = (NIMX - 1)*DELTAX - thickness_PML_x;
	for (int i = 1; i <= NIMX; i++) {
		xval = DELTAX*float(i - 1);
		abscissa_in_PML = xoriginleft - xval;//PML XMIN
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xoriginleft - (xval + DELTAX / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x_half[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xval - xoriginright;//PML XMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = xval + DELTAX / 2.0 - xoriginright;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_x;
			tempd_x_half[i] = d0_x*powf(abscissa_normalized, NPOWER);
			tempK_x_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_x_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}
		if (tempalpha_x[i] < 0.0) { tempalpha_x[i] = 0.0; }
		if (tempalpha_x_half[i] < 0.0) { tempalpha_x_half[i] = 0.0; }
		tempb_x[i] = expf(-(tempd_x[i] / tempK_x[i] + tempalpha_x[i])*DELTATT);
		tempb_x_half[i] = expf(-(tempd_x_half[i] / tempK_x_half[i] + tempalpha_x_half[i])*DELTATT);

		if (fabs(tempd_x[i]) > 1e-6) { tempa_x[i] = tempd_x[i] * (tempb_x[i] - 1.0) / (tempK_x[i] * (tempd_x[i] + tempK_x[i] * tempalpha_x[i])); }
		if (fabs(tempd_x_half[i]) > 1e-6) { tempa_x_half[i] = tempd_x_half[i] * (tempb_x_half[i] - 1.0) / (tempK_x_half[i] * (tempd_x_half[i] + tempK_x_half[i] * tempalpha_x_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_x, tempd_x, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_x_half, tempd_x_half, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_x, tempa_x, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_x_half, tempa_x_half, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_x, tempalpha_x, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_x_half, tempalpha_x_half, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_x, tempb_x, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_x_half, tempb_x_half, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_x, tempK_x, sizeof(float)*DIMX, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_x_half, tempK_x_half, sizeof(float)*DIMX, hipMemcpyHostToDevice));

	free(tempd_x); free(tempd_x_half); free(tempa_x); free(tempa_x_half); free(tempalpha_x); free(tempalpha_x_half); free(tempb_x); free(tempb_x_half); free(tempK_x); free(tempK_x_half);

	//-----------------PML Y
	float *tempd_y = (float*)malloc(sizeof(float)*DIMY);
	float *tempd_y_half = (float*)malloc(sizeof(float)*DIMY);
	float *tempa_y = (float*)malloc(sizeof(float)*DIMY);
	float *tempa_y_half = (float*)malloc(sizeof(float)*DIMY);
	float *tempb_y = (float*)malloc(sizeof(float)*DIMY);
	float *tempb_y_half = (float*)malloc(sizeof(float)*DIMY);
	float *tempK_y = (float*)malloc(sizeof(float)*DIMY);
	float *tempK_y_half = (float*)malloc(sizeof(float)*DIMY);
	float *tempalpha_y = (float*)malloc(sizeof(float)*DIMY);
	float *tempalpha_y_half = (float*)malloc(sizeof(float)*DIMY);

	for (int i = 1; i < DIMY; i++) {
		tempd_y[i] = 0.0;
		tempd_y_half[i] = 0.0;
		tempK_y[i] = 1.0;
		tempK_y_half[i] = 1.0;
		tempalpha_y[i] = 0.0;
		tempalpha_y_half[i] = 0.0;
		tempa_y[i] = 0.0;
		tempa_y_half[i] = 0.0;
		tempb_y[i] = 0.0;
		tempb_y_half[i] = 0.0;
	}

	yoriginbottom = thickness_PML_y;
	yorigintop = (NIMY - 1)*DELTAY - thickness_PML_y;
	for (int i = 1; i <= NIMY; i++) {
		yval = DELTAY*float(i - 1);
		abscissa_in_PML = yoriginbottom - yval;//PML YMIN
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yoriginbottom - (yval + DELTAY / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y_half[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yval - yorigintop;//PML YMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yval + DELTAY / 2.0 - yorigintop;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_y_half[i] = d0_y*powf(abscissa_normalized, NPOWER);
			tempK_y_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		tempb_y[i] = expf(-(tempd_y[i] / tempK_y[i] + tempalpha_y[i])*DELTATT);
		tempb_y_half[i] = expf(-(tempd_y_half[i] / tempK_y_half[i] + tempalpha_y_half[i])*DELTATT);

		if (fabs(tempd_y[i]) > 1e-6) { tempa_y[i] = tempd_y[i] * (tempb_y[i] - 1.0) / (tempK_y[i] * (tempd_y[i] + tempK_y[i] * tempalpha_y[i])); }
		if (fabs(tempd_y_half[i]) > 1e-6) { tempa_y_half[i] = tempd_y_half[i] * (tempb_y_half[i] - 1.0) / (tempK_y_half[i] * (tempd_y_half[i] + tempK_y_half[i] * tempalpha_y_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_y, tempd_y, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_y_half, tempd_y_half, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_y, tempa_y, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_y_half, tempa_y_half, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_y, tempalpha_y, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_y_half, tempalpha_y_half, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_y, tempb_y, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_y_half, tempb_y_half, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_y, tempK_y, sizeof(float)*DIMY, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_y_half, tempK_y_half, sizeof(float)*DIMY, hipMemcpyHostToDevice));

	free(tempd_y); free(tempd_y_half); free(tempa_y); free(tempa_y_half); free(tempalpha_y); free(tempalpha_y_half); free(tempb_y); free(tempb_y_half); free(tempK_y); free(tempK_y_half);

	//-----------------PML Z
	float *tempd_z = (float*)malloc(sizeof(float)*DIMZ);
	float *tempd_z_half = (float*)malloc(sizeof(float)*DIMZ);
	float *tempa_z = (float*)malloc(sizeof(float)*DIMZ);
	float *tempa_z_half = (float*)malloc(sizeof(float)*DIMZ);
	float *tempb_z = (float*)malloc(sizeof(float)*DIMZ);
	float *tempb_z_half = (float*)malloc(sizeof(float)*DIMZ);
	float *tempK_z = (float*)malloc(sizeof(float)*DIMZ);
	float *tempK_z_half = (float*)malloc(sizeof(float)*DIMZ);
	float *tempalpha_z = (float*)malloc(sizeof(float)*DIMZ);
	float *tempalpha_z_half = (float*)malloc(sizeof(float)*DIMZ);

	for (int i = 1; i < DIMZ; i++) {
		tempd_z[i] = 0.0;
		tempd_z_half[i] = 0.0;
		tempK_z[i] = 1.0;
		tempK_z_half[i] = 1.0;
		tempalpha_z[i] = 0.0;
		tempalpha_z_half[i] = 0.0;
		tempa_z[i] = 0.0;
		tempa_z_half[i] = 0.0;
		tempb_z[i] = 0.0;
		tempb_z_half[i] = 0.0;
	}

	zoriginbottom = thickness_PML_z;
	zorigintop = (NIMZ - 1)*DELTAZ - thickness_PML_z;
	for (int i = 1; i <= NIMZ; i++) {
		zval = DELTAZ*float(i - 1);
		abscissa_in_PML = zoriginbottom - zval;//PML ZMIN
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_y[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = yoriginbottom - (yval + DELTAY / 2.0);
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z_half[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = zval - zorigintop;//PML ZMAX
		if (abscissa_in_PML >= 0.0) {
			abscissa_normalized = abscissa_in_PML / thickness_PML_z;
			tempd_z[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		abscissa_in_PML = zval + DELTAZ / 2.0 - zorigintop;
		if (abscissa_in_PML >= 0.0){
			abscissa_normalized = abscissa_in_PML / thickness_PML_y;
			tempd_z_half[i] = d0_z*powf(abscissa_normalized, NPOWER);
			tempK_z_half[i] = 1.0 + (K_MAX_PML - 1.0)*powf(abscissa_normalized, NPOWER);
			tempalpha_z_half[i] = ALPHA_MAX_PML*(1.0 - abscissa_normalized) + 0.1*ALPHA_MAX_PML;
		}

		tempb_z[i] = expf(-(tempd_z[i] / tempK_z[i] + tempalpha_z[i])*DELTATT);
		tempb_z_half[i] = expf(-(tempd_z_half[i] / tempK_z_half[i] + tempalpha_z_half[i])*DELTATT);

		if (fabs(tempd_z[i]) > 1e-6) { tempa_z[i] = tempd_z[i] * (tempb_z[i] - 1.0) / (tempK_z[i] * (tempd_z[i] + tempK_z[i] * tempalpha_z[i])); }
		if (fabs(tempd_z_half[i]) > 1e-6) { tempa_z_half[i] = tempd_z_half[i] * (tempb_z_half[i] - 1.0) / (tempK_z_half[i] * (tempd_z_half[i] + tempK_z_half[i] * tempalpha_z_half[i])); }
	}

	HANDLE_ERROR(hipMemcpy(d_z, tempd_z, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_z_half, tempd_z_half, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_z, tempa_z, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(a_z_half, tempa_z_half, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_z, tempalpha_z, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(alpha_z_half, tempalpha_z_half, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_z, tempb_y, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(b_z_half, tempb_z_half, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_z, tempK_z, sizeof(float)*DIMZ, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(K_z_half, tempK_z_half, sizeof(float)*DIMZ, hipMemcpyHostToDevice));

	free(tempd_z); free(tempd_z_half); free(tempa_z); free(tempa_z_half); free(tempalpha_z); free(tempalpha_z_half); free(tempb_z); free(tempb_z_half); free(tempK_z); free(tempK_z_half);

	int *DDIMX, *DDIMY, *DDIMZ;
	HANDLE_ERROR(hipMalloc((void**)&DDIMX, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMY, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&DDIMZ, sizeof(int)));
	HANDLE_ERROR(hipMemcpy(DDIMX, &NIMX, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMY, &NIMY, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(DDIMZ, &NIMZ, sizeof(int), hipMemcpyHostToDevice));

	dim3 threads;
	threads.x = 100;
	threads.y = 100;
	threads.z = 100;

	dim3 blocks;
	blocks.x = NIMX / threads.x;
	blocks.y = NIMY / threads.y;
	blocks.z = NIMZ / threads.z;

	int *iit;
	HANDLE_ERROR(hipMalloc((void**)&iit, sizeof(int)));

	for (int it = 1; it <= NSTEP; it++) {
		kersigmaxyz << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ, memory_dvx_dx, memory_dvy_dy, memory_dvz_dz, a_x_half, a_y, a_z, b_x_half, b_y, b_z, K_x_half, K_y, K_z, DELTAT_lambdaplus2mu, DELTAT_lambda, sigmaxx, sigmayy, sigmazz, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vx, vy, vz);

		kersigmaxy << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ, memory_dvy_dx, memory_dvx_dy, a_x, a_y_half, b_x, b_y_half, K_x, K_y_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, vx, vy, DELTAT_mu, sigmaxy);

		kersigmaxzyz << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ,memory_dvz_dx, memory_dvx_dz, memory_dvz_dy, memory_dvy_dz, a_x, a_z, a_y_half, a_z_half, b_x, b_y_half, b_z_half, K_x, K_y_half, K_z_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vx, vy, vz, DELTAT_mu, sigmaxz, sigmayz);

		kervxvy << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ, sigmaxx, sigmaxy, sigmaxz, sigmayy, sigmayz, memory_dsigmaxx_dx, memory_dsigmaxy_dy, memory_dsigmaxz_dz, memory_dsigmaxy_dx, memory_dsigmayy_dy, memory_dsigmayz_dz, a_x, a_y, a_z, a_x_half, a_y_half, b_x, b_y, b_z, b_x_half, b_y_half, K_x, K_y, K_z, K_x_half, K_y_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, DELTAT_over_rho, vx, vy);

		kervz << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ, sigmaxz, sigmayz, sigmazz, memory_dsigmaxz_dx, memory_dsigmayz_dy, memory_dsigmazz_dz, b_x_half, b_y, b_z_half, a_x_half, a_y, a_z_half, K_x_half, K_y, K_z_half, ONE_OVER_DELTAX, ONE_OVER_DELTAY, ONE_OVER_DELTAZ, vz, DELTAT_over_rho);

		HANDLE_ERROR(hipMemcpy(iit, &it, sizeof(int), hipMemcpyHostToDevice));
		keraddSource << <blocks, threads >> >(iit, ISOURCE, JSOURCE, KSOURCE, ANGLE_FORCE, DEGREES_TO_RADIANS, DELTAT, factor, t0, ff0, DPI, vx, vy, rho);

		kerDirichletBoundary << <blocks, threads >> >(DDIMX, DDIMY, DDIMZ, vx, vy, vz);

		if (fmod(it, 200) == 0){
			float *tempvz = (float*)malloc(sizeof(float)*DIMX*DIMY*DIMZ);
			float *sxvz = (float*)malloc(sizeof(float)*NIMY*NIMZ);
			float *syvz = (float*)malloc(sizeof(float)*NIMX*NIMZ);
			float *szvz = (float*)malloc(sizeof(float)*NIMX*NIMY);
			HANDLE_ERROR(hipMemcpy(tempvz, vz, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyDeviceToHost));

			//slicing for snapshot
			for (int k = 1; k <= NIMZ; k++) {
				for (int j = 1; j <=NIMY; j++) {
					int jk = (j-1) + (k-1)*NIMY;
					int ijk = ISOURCEE + j*NIMX + k*NIMX*NIMY;
					sxvz[jk] = tempvz[ijk];
				}

				for (int i = 1; i <= NIMX - 1; i++) {
					int ik = (i-1) + (k-1)*NIMX;
					int ijk = i + JSOURCEE*NIMX + k*NIMX*NIMY;
					syvz[ik] = tempvz[ijk];
				}
			}
			for (int j = 1; j <= NIMY; j++) {
				for (int i = 1; i <= NIMX; i++) {
					int ij = (i-1) + (j-1)*NIMX;
					int ijk = i + j*NIMX + KSOURCEE * NIMX*NIMY;
					szvz[ij] = tempvz[ijk];
				}
			}
			//save to file
			char nmfile1[50]; char nmfile2[50]; char nmfile3[50];
			sprintf_s(nmfile1, "sxvz%05i.sxvz", it);
			sprintf_s(nmfile2, "syvz%05i.syvz", it);
			sprintf_s(nmfile3, "szvz%05i.szvz", it);
			errno_t err;
			FILE *file1, *file2, *file3;
			err = fopen_s(&file1, nmfile1, "wb");
			if (err == 0) {
				printf("Capturing sxvz %05i \n", it);
				for (int k = 0; k < NIMZ; k++) {
					for (int j = 0; j < NIMY; j++) {
						int jk = j + k*NIMY;
						float f1 = sxvz[jk];
						fwrite(&f1, sizeof(float), 1, file1);
					}
				}
				fclose(file1);
			}
			err = fopen_s(&file2, nmfile2, "wb");
			if (err == 0) {
				printf("Capturing syvz %05i \n", it);
				for (int k = 0; k < NIMZ; k++) {
					for (int i = 0; i < NIMX; i++) {
						int ik = i + k*NIMX;
						float f2 = syvz[ik];
						fwrite(&f2, sizeof(float), 1, file2);
					}
				}
				fclose(file2);
			}
			err = fopen_s(&file3, nmfile3, "wb");
			if (err == 0) {
				printf("Capturing szvz %05i \n", it);
				for (int j = 0; j < NIMY; j++) {
					for (int i = 0; i < NIMX; i++) {
						int ij = i + j*NIMX;
						float f3 = szvz[ij];
						fwrite(&f3, sizeof(float), 1, file3);
					}
				}
				fclose(file3);
			}
			_fcloseall();
			//save to file END
			free(tempvz);
			free(sxvz); free(syvz); free(szvz);
		}
	}

	HANDLE_ERROR(hipFree(sigmaxx)); HANDLE_ERROR(hipFree(sigmayy)); HANDLE_ERROR(hipFree(sigmazz));
	HANDLE_ERROR(hipFree(sigmaxy)); HANDLE_ERROR(hipFree(sigmaxz)); HANDLE_ERROR(hipFree(sigmayz));
	HANDLE_ERROR(hipFree(memory_dsigmaxx_dx)); HANDLE_ERROR(hipFree(memory_dsigmaxy_dx)); HANDLE_ERROR(hipFree(memory_dsigmaxy_dy));
	HANDLE_ERROR(hipFree(memory_dsigmaxz_dx)); HANDLE_ERROR(hipFree(memory_dsigmaxz_dz)); HANDLE_ERROR(hipFree(memory_dsigmayy_dy));
	HANDLE_ERROR(hipFree(memory_dsigmayz_dy)); HANDLE_ERROR(hipFree(memory_dsigmayz_dz)); HANDLE_ERROR(hipFree(memory_dsigmazz_dz));
	HANDLE_ERROR(hipFree(memory_dvx_dx)); HANDLE_ERROR(hipFree(memory_dvx_dy)); HANDLE_ERROR(hipFree(memory_dvx_dz));
	HANDLE_ERROR(hipFree(memory_dvy_dx)); HANDLE_ERROR(hipFree(memory_dvy_dy)); HANDLE_ERROR(hipFree(memory_dvy_dz));
	HANDLE_ERROR(hipFree(memory_dvz_dx)); HANDLE_ERROR(hipFree(memory_dvz_dy)); HANDLE_ERROR(hipFree(memory_dvz_dz));
	return 0;
}
